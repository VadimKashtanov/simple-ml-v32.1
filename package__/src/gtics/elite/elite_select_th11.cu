#include "hip/hip_runtime.h"
#include "package/head/gtics/elite/elite.cuh"

static __global__
void kernel_elite_select(
	uint sets, uint ws,
	uint elites, uint portion,
	uint seed,
	float * _old, float * _new)
{
	uint th_x = threadIdx.x + blockIdx.x * blockDim.x;
	uint th_elite = blockIdx.y;

	uint elite_pos, clone_pos;

	if (thx < ws) {
	
		for (uint i=0; i < portion; i++)
		{
			//	Make a new clone and add weights

#define PODIUM ((uint*)const_mem)	/*const_mem is `float*`, so we have to interpret it as `uint*` */

			elite_pos = PODIUM[th_elite]*ws + th_x;
			clone_pos = PODIUM[th_elite + th_elite*portion + i]*ws + th_x;

#undef PODIUM

			_new[clone_pos] = _old[elite_pos] + pseudo_randomf(seed + clone_pos) * 0.1;
		}
	}
}

void gtic_select_elite(Gtic_t * gtic) {
	float * new_weights_d;

	uint sets = gtic->opti->sets;
	uint ws = gtic->opti->mdl->weights;

	SAFE_CUDA(hipMalloc((void**)&new_weights_d, sizeof(float) * ws * sets));

	//	########### Build podium [best-set-id, 2nd-best-set-id, ... worst-set-id]
	SAFE_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(const_mem), gtic->opti->podium, len * sizeof(uint)));	//const_mem is float* type, but it's not a probleme because type doesn't matters, you can (uint*)const_meme, and it compute as an `uint` and not `float`. Float is juste a size and a data structure.
	
	//	########## Launch Cloning
	kernel_elite_select<<<dim3(KERN_DIV(ws,16), elites),dim3(16,1)>>>(
		sets, ws,
		elites, portion,
		rand() % 100000,
		gtic->opti->train->_weight, new_weights_d,
	);

	//	############ Copy new to old in `Train_t` and free the tempt `new_weights_d`
	SAFE_CUDA(hipMemcpy(gtic->opti->train->_weight, new_weights_d, sizeof(float) * ws * sets, hipMemcpyDeviceToDevice));
	SAFE_CUDA(hipFree(new_weights_d));
};