#include "hip/hip_runtime.h"
#include "pkg_head/insts/gaussfiltre1d/gaussfiltre1d_th11.cuh"

__global__
void gaussfiltre1d_forward_th1x1(
	uint len,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd,
	uint seed,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	float _tmp;

	if (x < len) {
		_tmp = var[time*sets*total + istart + x] + weight[ws*set + wstart + x];
		var[time*sets*total + set*total + ystart + x] = exp(-pow(_tmp,2));
		locd[time*sets*lsize + set*lsize + lstart + x] = -2*_tmp*exp(-pow(_tmp,2));
	}
};