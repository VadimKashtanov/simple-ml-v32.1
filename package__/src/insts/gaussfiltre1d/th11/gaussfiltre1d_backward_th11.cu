#include "hip/hip_runtime.h"
#include "pkg_head/insts/gaussfiltre1d/gaussfiltre1d_th11.cuh"

__global__
void gaussfiltre1d_backward_th1x1(
	uint len,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	float dlds;

	if (x < len) {
		dlds = grad[time*sets*total + set*total + ystart + x] * locd[time*sets*lsize + set*lsize + lstart + x];

		grad[time*sets*total + istart + x] += dlds;
		meand[ws*set + wstart + x] += dlds;
	}
};