#include "hip/hip_runtime.h"
#include "pkg_head/insts/gaussfiltre1d/gaussfiltre1d_th11.cuh"

__global__
void gaussfiltre1d_use_th1x1(
	uint len,
	uint time,
	uint total,
	uint istart, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;

	if (x < len) {
		var[time*total + ystart + x] = exp(-pow(var[time*total + istart + x] + weight[wstart + x],2));
	}
};