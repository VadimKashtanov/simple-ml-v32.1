#include "hip/hip_runtime.h"
#include "pkg_head/insts/gaussfiltre1d/gaussfiltre1d.cuh"

//			   0      1      2      3      4 
//Arguments = [len, istart,ystart,wstart,lstart]

void gaussfiltre1d_check(uint * param) {
	if (param[0] == 0) raise(SIGINT);
};

void gaussfiltre1d_cpu(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint len=mdl->param[inst][0],		\
		 istart=mdl->param[inst][1],	\
		 ystart=mdl->param[inst][2],	\
		 wstart=mdl->param[inst][3],	\
		 locdstart=mdl->param[inst][4];

	uint inp = total*time + istart;
	uint out = total*time + ystart;

	uint total = mdl->total;

	float * var = cpu->var;
	float * weight = mdl->weight;

	float a,p;

	for (uint y=0; y < len; y++) {
		a = var[inp + (y)];
		p = weight[wstart + (x)];
		var[out + y] = exp(-(a+p)**2);
	}
};

void gaussfiltre1d_use(Use_t * use, uint inst, uint time) {
	gaussfiltre1d_use_call_mode_th11(use, inst, time);
};

void gaussfiltre1d_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	gaussfiltre1d_forward_call_mode_th11(train, inst, time, start_seed);	
};

void gaussfiltre1d_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	gaussfiltre1d_backward_call_mode_th11(train, inst, time, start_seed);
};