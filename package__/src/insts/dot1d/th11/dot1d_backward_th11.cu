#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot1d.cuh"

__global__
void dot1d_backward_th1x1(
	uint Ax, uint Yx,
	uint activ,
	uint time,
	uint input_start, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	/*	Kernel coordinates	*/
	uint _Yx = threadIdx.x + blockIdx.x*blockDim.x, \
		 set = blockIdx.y;

	if (_Yx < Yx) {
		uint Apos = time*sets*total + set*total + input_start;
		uint weight_start = set*wsize + wstart;
		uint Bpos = weight_start + _Yx*Ax;

		float dlds = locd[time*sets*locdsize + set*locdsize + locdstart + _Yx] * grad[time*sets*total + set*total + ystart + _Yx];

		meand[weight_start + Yx*Ax + _Yx] += dlds;

		for (uint i=0; i < Ax; i++) {
			if (pseudo_randomf(Apos*seed) >= drop_rate) {
				atomicAdd(&grad[Apos], dlds * weight[Bpos]);
				atomicAdd(&meand[Bpos], dlds * var[Apos]);
			}
			Apos++;
			Bpos += Yx;
		}
	}
};