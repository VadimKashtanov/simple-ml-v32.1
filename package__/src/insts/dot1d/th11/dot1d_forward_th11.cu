#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot1d.cuh"

__global__
void dot1d_forward_th1x1(
	uint Ax, uint Yx,
	uint activ,
	uint time,
	uint input_start, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd,
	uint seed, float drop_rate,
	uint sets)
{
	uint _Yx = threadIdx.x + blockIdx.x*blockDim.x, \
		 set = blockIdx.y;

	if (_Yx < Yx) {

		uint Apos = time*sets*total + set* + input_start;
		uint weight_start = set*wsize + wstart;
		uint Bpos = weight_start + _Yx*Ax;

		float sum = 0;
		for (uint i=0; i < Ax; i++) {
			if (pseudo_randomf(Apos*seed) >= drop_rate)
				sum += var[Apos] * weight[Bpos];
			Apos++;
			Bpos += Yx;
		}
		sum += weight[weight_start + Yx*Ax + _Yx];
		
		float __locd;

		if (activ == 0) {
			sum = 1 / (1 + exp(-sum));
			__locd = sum*(1 - sum);	//f'(x) = f(x)(1 - f(x))
		} else if (activ == 1) {
			sum = tanh(sum);
			__locd = 1 - sum*sum;	//f'(x) = 1 - tanh(x)^2
		} else if (activ == 2) {
			__locd = sum;
			sum = exp(-pow(sum,2));
			__locd = -2*__locd*sum;	//f'(x) = -2x*e^(-x^2)
		} else  if (activ == 3) {
			__locd = (sum > 0);
			sum = sum*__locd;
		}

		var[time*sets*total + set*total + ystart + _Yx] = sum;		//same assembler than putting it in if/else structure
		locd[time*sets*locdsize + set*locdsize + locdstart + _Yx] = __locd;
	}
};
