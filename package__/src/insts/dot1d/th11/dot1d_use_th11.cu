#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot1d.cuh"

__global__
void dot1d_use_th1x1(
	uint Ax, uint Yx,
	uint activ,
	uint time,
	uint total,
	uint input_start, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint _Yx = threadIdx.x + blockIdx.x*blockDim.x;

	if (_Yx < Yx) {
		uint Apos = time*total + input_start;
		uint Bpos = wstart + _Yx*Ax;	//	Dot1d does not store W as Dot2d       in fact Dot2D.T = Dot1d  (it would be better to change it)

		float sum = 0;
		for (uint i=0; i < Ax; i++) {
			sum += var[Apos] * weight[Bpos];
			Apos++;
			Bpos += Yx;
		}
		sum += weight[wstart + Yx*Ax + _Yx];

		if (activ == 0) sum = 1 / (1 + exp(-sum));
		else if (activ == 1) sum = tanh(sum);
		else if (activ == 2) sum = exp(-pow(sum,2));
		else sum = sum*(sum > 0);

		var[time*total + ystart + _Yx] = sum;
	}
};