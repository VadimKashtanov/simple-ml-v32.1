#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot1d/dot1d.cuh"

void dot1d_check(uint * param) {
	//>0 <==> >= 1
	if (param[0] == 0) raise(SIGINT);
	if (param[1] == 0) raise(SIGINT);
	if (param[7] >100) raise(SIGINT);
	if (param[2] >= 4) raise(SIGINT);
};

void dot1d_cpu(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint Ax=mdl->param[inst][0],				\
		 Yx=mdl->param[inst][1],				\
		 activ=mdl->param[inst][2],			\
		 input_start=mdl->param[inst][3],		\
		 ystart=mdl->param[inst][4],			\
		 wstart=mdl->param[inst][5];

	float * var = cpu->var;
	float * weight = mdl->weight;

	float _tmp;
	uint _inp=time*mdl->total + input_start,	\
		 _w=wstart;
	
	for (uint y=0; y < Yx; y++) {
		_tmp = 0;
		//Scalar product of 2 vectors in input (A) and weight (B)
		for (uint i=0; i < Ax; i++)
			_tmp += var[_inp + i] * weight[_w + y*Yx + i];//weight[_w + i*Yx];

		//Adding bias
		_tmp += weight[wstart + Ax*Yx + x];
		
		//Activation
		if (activ == 0) _tmp = 1 / (1 + exp(-_tmp));
		else if (activ == 1) _tmp = tanh(_tmp);
		else if (activ == 2) _tmp = exp(-_tmp*_tmp);
		else _tmp = _tmp * (_tmp > 0);
		
		//Write it to Y
		var[time*mdl->total + ystart + y] = _tmp;

		//Next colon of weights
		//_w++;
	}
};

void dot1d_use(Use_t * use, uint inst, uint time) {
	dot1d_use_call_mode_th11(use, inst, time);
};

void dot1d_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	dot1d_forward_call_mode_th11(train, inst, time, start_seed);
};

void dot1d_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	dot1d_backward_call_mode_th11(train, inst, time, start_seed);
};