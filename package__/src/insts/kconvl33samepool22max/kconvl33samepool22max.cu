#include "hip/hip_runtime.h"
#include "pkg_head/insts/kconvl33samepool22max/kconvl33samepool22max.cuh"

void kconvl33samepool22max_check(uint * param) {
	if (param[0] == 0) raise(SIGINT);
	if (param[1] == 0) raise(SIGINT);
	if (param[0]%2!=0) raise(SIGINT);
	if (param[1]%2!=0) raise(SIGINT);
	if (param[2] == 0) raise(SIGINT);
	if (param[3] == 0) raise(SIGINT);
	if (param[4] >= 4) raise(SIGINT);
	if (param[9] >100) raise(SIGINT);
}

static float max_4(float _00, float _10, float _01, float _11) {
	float max = _00;
	if (max < _10) max = _10;
	if (max < _01) max = _01;
	if (max < _11) max = _11;
	return max;
};

static float activate(float x, uint activ) {
	if (activ == 0) return 1 / (1 + exp(-x));
	else if (activ == 1) return tanh(x);
	else if (activ == 2) return exp(-x*x);
	else return x * (x >= 0);
};

void kconvl33samepool22max_cpu_call(
	Cpu_t * cpu, uint inst, uint time)
{
	Mdl_t * mdl = cpu->mdl;

	uint Ax=mdl->param[inst][0],		\
		 Ay=mdl->param[inst][1],		\
		 n0=mdl->param[inst][2],			\
		 n1=mdl->param[inst][3],			\
		 activ=mdl->param[inst][4],		\
		 istart=mdl->param[inst][5],	\
		 ystart=mdl->param[inst][6],		\
		 wstart=mdl->param[inst][7];

	float * var = cpu->var;
	float * weight = mdl->weight;

	uint Yx=Ax/2, Yy=Ay/2;

	float _00, _01, _10, _11;
	float __w;
	uint bias;

	int __y, __x;

	for (uint _n1=0; _n1 < n1; _n1++) {
		for (uint y=0; y < Yy; y++) {
			for (uint x=0; x < Yx; x++) {
				_00 = 0;
				_10 = 0;
				_01 = 0;
				_11 = 0;
				for (uint _n0=0; _n0 < n0; _n0++) {
					for (uint _x=0; _x < 3; _x++) {
						for (uint _y=0; _y < 3; _y++) {
							__w = weight[wstart + _n1*9*n0 + _n0*9 + _y*3 + _x];

							//(0,0)
							__y = y*2+_y-1;
							__x = x*2+_x-1;
							if (__y >=0 && __x >= 0)
								_00 += var[time*mdl->total + istart + __y*Ax + __x] * __w;

							//(1,0)
							__y = y*2+_y-1;
							__x = x*2+_x;
							if (__y >= 0 && __x < Ax)
								_10 += var[time*mdl->total + istart + __y*Ax + __x] * __w;

							//(0,1)
							__y = y*2+_y;
							__x = x*2+_x-1;
							if (__y < Ay && __x >= 0)
								_01 += var[time*mdl->total + istart + __y*Ax + __x] * __w;

							//(1,1)
							__y = y*2+_y;
							__x = x*2+_x;
							if (__y < Ay && __x < Ax)
								_11 += var[time*mdl->total + istart + __y*Ax + __x] * __w;
						}
					}
				}
				
				bias = wstart + 9*n1*n0 + _n1*Ax*Ay;
				_00 = activate(_00 + weight[bias + y*2*Ax + x*2], activ);			//bias tensor is same size as X, because kconvl is same size as X
				_10 = activate(_10 + weight[bias + y*2*Ax + x*2 + 1], activ);		//bias is added to .k
				_01 = activate(_01 + weight[bias + (y*2+1)*Ax + x*2], activ);		//and after that polled
				_11 = activate(_11 + weight[bias + (y*2+1)*Ax + x*2 + 1], activ);	//

				//
				var[time*mdl->total + ystart + _n1*Yx*Yy + y*Yx + x] = max_4(_00, _10, _01, _11);
			}
		}
	}
}

void kconvl33samepool22max_use_call(Use_t * use, uint inst, uint time) {
	Mdl_t * mdl = use->mdl;
	
	uint n0=mdl->param[inst][2];
	uint n1=mdl->param[inst][3];

	if (n0*n1*9 < MAX_CONST_FLOATS) {
		kconvl33samepool22max_use_call_mode_th11(use, inst, time);
	} else {
		ERR("n0*n1*9 is more than const meme.");
	}
}

void kconvl33samepool22max_forward_call(Train_t * train, uint inst, uint time, uint start_seed)
{
	Mdl_t * mdl = train->mdl;

		//Xxlen=mdl->param[inst][0],			
		//Xylen=mdl->param[inst][1],			
	uint n0=mdl->param[inst][2];
	uint n1=mdl->param[inst][3];				
		// activ=mdl->param[inst][4],			
		// input_start=mdl->param[inst][5];
		// ystart=mdl->param[inst][6];		
		// wstart=mdl->param[inst][7];			
		// locdstart=mdl->param[inst][8];		
		// drop_rate_int=mdl->param[inst][9];

	//uint Yxlen = Xxlen/2,	\
	//	 Yylen = Xylen/2;

	if (n0*n1*9 < MAX_CONST_FLOATS) {
		kconvl33samepool22max_forward_call_mode_th11(train, inst, time, start_seed);
		//forward_const_th11(train, inst, time, start_seed);
	} else {
		ERR("n0*n1*9 is more than const meme.");
	}
};

void kconvl33samepool22max_backward_call(Train_t * train, uint inst, uint time, uint start_seed)
{
	Mdl_t * mdl = train->mdl;

		//Xxlen=mdl->param[inst][0],			
		//Xylen=mdl->param[inst][1],			
	uint n0=mdl->param[inst][2];
	uint n1=mdl->param[inst][3];				
		// activ=mdl->param[inst][4],			
		// input_start=mdl->param[inst][5];
		// ystart=mdl->param[inst][6];		
		// wstart=mdl->param[inst][7];			
		// locdstart=mdl->param[inst][8];		
		// drop_rate_int=mdl->param[inst][9];

	//uint Yxlen = Xxlen/2,	\
	//	 Yylen = Xylen/2;

	if (n0*n1*9 < MAX_CONST_FLOATS) {
		kconvl33samepool22max_backward_call_mode_th11(train, inst, time, start_seed);
	} else {
		ERR("n0*n1*9 is more than const meme.");
	}
};