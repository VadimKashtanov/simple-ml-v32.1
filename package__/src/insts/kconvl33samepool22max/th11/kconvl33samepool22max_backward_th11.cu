#include "hip/hip_runtime.h"
#include "pkg_head/insts/kconvl33samepool22max.cuh"

__global__
void kconvl33samepool22max_backward_const_th1x1(
	uint n0, uint n1, uint Ax, uint Ay,
	uint activ,
	uint time,
	uint total, uint wsize, uint lsize,
	uint istart, uint wstart, uint ystart, uint lstart,
	float * var, float * weight, float * locd,
	float * grad, float * meand,
	uint seed, float drop_rate,
	uint set, uint sets)
{
	/*
		On divise la grille avec 1 kernel par pixel.
		Vu que chaqu'un des pixels fait pool22max, il y un block avec 4 _xx de convolution.
		la derivee local de y = max(a,b,c,d) en a,b,c,d est null pour ceux qui ne sont pas max.
		Donc au finale on calcule que pour un seul _xx (ce pixel est dans `.k = .x | .K`).
		A partire de la on peut considere que max existe plus, vu que c'est pour lui la fonction (lambda x:x), donc c'est juste une copie.
		Donc la y = activation(_xx + bias, activ). Donc dL/d_xx += dL/dY * dY/d(activation) * d(activation)/d_xx
		Un peut comme dans dot2d on ajoute a d(pixel maximum):
			dL/dY 				c'est juste l'erreur, qui sera dans un gros model, juste le gradient de l'input de l'instruction suivante
			dY/d(activation) 	c'est la dérivée local : locd[0]  (ou locd[1] est le max_id), chaque pixel de Y a 2 locd
			d(activation)/d_xx 	c'est ducoup le kernel (vu que c'est une simple multiplication)

		la derivee du bias c'est just dL/d_xx += dL/dY * dY/d(activation)
		car d(activation)/dbias == 1  (y = x + 1*b => dy/db = 1)

		Ici l'algorithm fait juste un ajustement avec (y,x) et (pool_block_x,pool_block_y) pour se mettre sur le bon pixel maximum
	*/
	uint out_x = threadIdx.x + blockIdx.x*blockDim.x,	\	//+1 because we don't compute border of output
		 out_y = threadIdx.y + blockIdx.y*blockDim.y;		//+1 it's an usefull approximation
	uint _n1   = threadIdx.z + blockIdx.z*blockDim.z;

	if (out_x < Ax/2 && out_y < Ay/2 && _n1 < n1) {
		uint _Ax = out_x * 2;
		uint _Ay = out_y * 2;

		//	Debut du kernel et de l'image
		uint kstart = _n1*9*n0;
		uint K = set*wsize + wstart;
		uint istart = time*sets*total + set*total + istart;

		//	On load les local derivee et le pixel maximum
		uint this_y_pixel_locd = time*sets*lsize + set*lsize + lstart + _n1*2*(Ax*Ay/4) + out_y*2*(Ax/2) + 2*out_x;
		float __locd = 	locd[this_y_pixel_locd];
		uint max_id = (uint)locd[this_y_pixel_locd + 1];

		//	Erreur * derivee local
		float dLdS = grad[time*sets*total + set*total + ystart + _n1*(Ax*Ay/4) + out_y*(Ax/2) + out_x] * __locd;
		
		uint imgpos;
		int y,x;

		int pool_block_x=max_id%2, pool_block_y=(max_id - max_id%2)/2;

		uint bias = set*wsize + wstart + n0*n1*9 + _n1*Ax*Ay + _Ay*Ax + _Ax;
		meand[bias + pool_block_y*Ax + pool_block_x] += dLdS;

		for (uint _n0=0; _n0 < n0; _n0++) {
			for (int i=-1; i < 2; i++) {
				for (int j=-1; j < 2; j++) {
					y = _Ay + i + pool_block_y;
					x = _Ax + j + pool_block_x;

					imgpos = istart + _n0*Ax*Ay + y*Ax + x;

					if (pseudo_randomf(imgpos + seed) >= drop_rate && y >= 0 && x >= 0 && y < Ay && x < Ax) {
						atomicAdd(&grad[imgpos], 		dLdS * const_mem[kstart]);
						atomicAdd(&meand[K + kstart], 	dLdS * var[imgpos]);
					}

					kstart++;
				}

			}
		}
	}
};