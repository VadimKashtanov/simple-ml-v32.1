#include "hip/hip_runtime.h"
#include "pkg_head/insts/kconvl33samepool22max.cuh"

void kconvl33samepool22max_train_const_MemCpyToSymbol(float * arr, uint len) {
	SAFE_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(
		const_mem),
		arr,
		len * sizeof(float)
	))
};			//hipMemcpyToSymbol have to be in same file, as kernels that will use this const_mem
			//__constant__ have to be declared in one .cuh
			// pourquoi ?
			// nvcc pas parfait

static __device__
float activate(float x, uint activ) {
	if (activ == 0) return 1 / (1 + exp(-x));
	else if (activ == 1) return tanh(x);
	else if (activ == 2) return exp(-x*x);
	else return x * (x >= 0);
};

static __device__
float max_of_4(float a, float b, float c, float d, float * max_id) {
	uint _max_id = 0;
	float max = a;
	if (b > a) {
		max = b;
		_max_id = 1; 
	}
	if (c > max) {
		max = c;
		_max_id = 2;
	}
	if (d > max) {
		max = d;
		_max_id = 3;
	}
	*max_id = _max_id;
	return max;
};

static __device__
float compute_locd(float a, float x, uint activ) {
	if (activ == 0) return a * (1 - a);
	else if (activ == 1) return 1 - a*a;
	else if (activ == 2) return -2*x*a;
	else return x >= 0;
};

__global__
void kconvl33samepool22max_forward_const_th1x1(
	uint n0, uint n1, uint Ax, uint Ay,
	uint activ,							
	uint time,
	uint total, uint wsize, uint lsize,
	uint istart, uint wstart, uint ystart, uint lstart,
	uint seed, float drop_rate,
	uint set, uint sets,
	float * var, float * weight, float * locd)
{
	uint out_x = threadIdx.x + blockIdx.x*blockDim.x,	\	//+1 because we don't compute border of output
		 out_y = threadIdx.y + blockIdx.y*blockDim.y;		//+1 it's an usefull approximation
	uint _n1   = threadIdx.z + blockIdx.z*blockDim.z;

	if (out_x < Ax/2 && out_y < Ay/2 && _n1 < n1) {

		int _Ax = out_x * 2;
		int _Ay = out_y * 2;

		uint istart = time*sets*total + set*total + istart;

		float _00=0, _10=0, _01=0, _11=0;

		float _image_value;

		int ximg, yimg, imgpos;
		uint kpos;

		for (uint _n0=0; _n0 < n0; _n0++) {
			//Iterate thought All accesible pixels from the _00,_10,_01,_11
			for (uint y=0; y < 4; y++) {
				for (uint x=0; x < 4; x++) {
					yimg = _Ay+y-1;
					ximg = _Ax-1+x;
					imgpos = istart + _n0*Ax*Ay + yimg*Ax + ximg;

					if (ximg>=0 && ximg<Ax && yimg>=0 && yimg<Ay && pseudo_randomf(imgpos + seed) >= drop_rate) {
						kpos = _n1*9*n0 + _n0*9 + y*3 + x;

						_image_value = var[imgpos];
						
						if (_image_value != 0.0) {
							if (x<3 && y<3)
								_00 += _image_value * const_mem[kpos-0-0];
							
							if (x>0 && y<3)
								_10 += _image_value * const_mem[kpos-1-0];
							
							if (x<3 && y>0)
								_01 += _image_value * const_mem[kpos-0-3];
							
							if (x>0 && y>0)
								_11 += _image_value * const_mem[kpos-1-3];
						}
					}
				}
			}
			/*for (int i=-1; i < 2; i++) {
				for (int j=-1; j < 2; j++) {
					_image_value = const_mem[_n1*9*n0 + _n0*9 + (i+1)*3 + (j+1)];
					if (_Ax + j >= 0 && _Ay + i >= 0)
						_00 += _image_value * var[istart + _n0*Ax*Ay + (_Ay + i)*Ax + (_Ax + j)];

					if (_Ax + j + 1 < Ax && _Ay + i >= 0 )
						_10 += _image_value * var[istart + _n0*Ax*Ay + (_Ay + i)*Ax + (_Ax + j + 1)];

					if (_Ax + j >=0 && _Ay + i + 1 < Ay)
						_01 += _image_value * var[istart + _n0*Ax*Ay + (_Ay + i + 1)*Ax + (_Ax + j)];

					if (_Ax + j + 1 < Ax && _Ay + i + 1 < Ay)
						_11 += _image_value * var[istart + _n0*Ax*Ay + (_Ay + i + 1)*Ax + (_Ax + j + 1)];
				}
			}*/
		}

		//	Bias
		uint bias = set*wsize + wstart + n0*n1*9 + _n1*Ax*Ay + _Ay*Ax + _Ax;

		_00 += weight[bias   	   ];
		_10 += weight[bias + 1 	   ];
		_01 += weight[bias + Ax	   ];
		_11 += weight[bias + Ax + 1];

		float __locd, max, max_id;
		float a_00, a_10, a_01, a_11;

		a_00 = activate(_00, activ);
		a_10 = activate(_10, activ);
		a_01 = activate(_01, activ);
		a_11 = activate(_11, activ);

		max = max_of_4(a_00, a_10, a_01, a_11, &max_id);

		if (max_id == 0) __locd = compute_locd(a_00, _00, activ);
		else if (max_id == 1) __locd = compute_locd(a_10, _10, activ);
		else if (max_id == 2) __locd = compute_locd(a_01, _01, activ);
		else if (max_id == 3) __locd = compute_locd(a_11, _11, activ);

		//printf("%f, %f, %f, %f = %f\n", a_00, a_10, a_01, a_11, max_id);

		//									*2 is because size is output*2 (__locd, max_id)
		uint this_y_pixel_locd = time*sets*lsize + set*lsize + lstart + _n1*2*(Ax*Ay/4) + out_y*2*(Ax/2) + 2*out_x;

		locd[this_y_pixel_locd	  ] = __locd;
		locd[this_y_pixel_locd + 1] = max_id;

		var[time*sets*total + set*total + ystart + _n1*(Ax*Ay/4) + out_y*(Ax/2) + out_x] = max;
	}
};