#include "hip/hip_runtime.h"
#include "pkg_head/insts/kconvl33samepool22max.cuh"

void kconvl33samepool22max_use_const_MemCpyToSymbol(float * arr, uint len) {
	SAFE_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(
		const_mem), 
		arr,
		len * sizeof(float)
	))
};

__device__ static
inline float max_of_4(float a, float b, float c, float d) {
	float max = a;
	if (b > a) max = b;
	if (c > max) max = c;
	if (d > max) max = d;
	return max;
};

__global__
void kconvl33samepool22max_use_const_th1x1(
	uint n0, uint n1, uint Ax, uint Ay,
	uint activ,							
	uint time,
	uint total, uint wsize,
	uint istart, uint wstart, uint ystart,
	float * var, float * weight)
{
	uint out_x = threadIdx.x + blockIdx.x*blockDim.x,	\	//+1 because we don't compute border of output
		 out_y = threadIdx.y + blockIdx.y*blockDim.y;		//+1 it's an usefull approximation
	uint _n1   = threadIdx.z + blockIdx.z*blockDim.z;

	//ou out_x <= Ax/2-1
	if (out_x < Ax/2 && out_y < Ay/2) {	//car il y aura des truc en trop (normalement c'est divisible par 2)
		uint _Ax = out_x * 2;
		uint _Ay = out_y * 2;

		uint kstart = _n1*9*n0;
		uint istart = time*total + istart;

		float _00=0, _10=0, _01=0, _11=0;
		uint cond;

		float _image_value;

		int ximg, yimg;
		uint kpos;

		for (uint _n0=0; _n0 < n0; _n0++) {
			//Iterate thought Image
			for (uint y=0; y < 4; y++) {
				for (uint x=0; x < 4; x++) {
					yimg = _Ay+y-1;
					ximg = _Ax-1+x;

					if (ximg>=0 && ximg<Ax && yimg>=0 && yimg<Ay) {
						kpos = kstart + _n0*9 + y*3 + x;

						_image_value = var[istart + _n0*Ax*Ay + yimg*Ax + ximg];

						if (_image_value != 0.0) {
							//
							cond = x<3 && y<3;
							//_00 += _image_value * const_mem[cond*(kpos)]*cond;
							if (cond) {
								_00 += _image_value * const_mem[kpos];
							}
							
							cond = x>0 && y<3;
							//_10 += _image_value * const_mem[cond*(kpos-1)]*cond;
							if (cond) {
								_10 += _image_value * const_mem[kpos-1];
							}
							
							cond = x<3 && y>0;
							//_01 += _image_value * const_mem[cond*(kpos-3)]*cond;
							if (cond) {
								_01 += _image_value * const_mem[kpos-3];
							}
							
							cond = x>0 && y>0;
							//_11 += _image_value * const_mem[cond*(kpos-4)]*cond;
							if (cond) {
								_11 += _image_value * const_mem[kpos-4];
							}
						}
					}
				}
			}

			//
			//istart = _n0*Ax*Ay;
			//istart += Ax*Ay;
			//kstart += 9;
		}

		//	Bias
		uint bias = wstart + n0*n1*9 + _n1*Ax*Ay + _Ay*Ax + _Ax;

		_00 += weight[bias   	   ];
		_10 += weight[bias + 1 	   ];
		_01 += weight[bias + Ax	   ];
		_11 += weight[bias + Ax + 1];

		if (activ == 0) {
			_00 = 1 / (1 + exp(-_00));
			_10 = 1 / (1 + exp(-_10));
			_01 = 1 / (1 + exp(-_01));
			_11 = 1 / (1 + exp(-_11));
		} else if (activ == 1) {
			_00 = tanh(_00);
			_10 = tanh(_10);
			_01 = tanh(_01);
			_11 = tanh(_11);
		} else if (activ == 2) {
			_00 = exp(-_00*_00);
			_10 = exp(-_10*_10);
			_01 = exp(-_01*_01);
			_11 = exp(-_11*_11);
		} else {
			_00 = _00*(_00 >= 0);
			_10 = _10*(_10 >= 0);
			_01 = _01*(_01 >= 0);
			_11 = _11*(_11 >= 0);
		}

		var[time*total + ystart + _n1*(Ax*Ay/4) + out_y*(Ax/2) + out_x] = max_of_4(_00, _10, _01, _11);
	}
};