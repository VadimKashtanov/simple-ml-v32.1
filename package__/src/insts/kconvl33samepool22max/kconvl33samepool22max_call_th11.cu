#include "pkg_head/insts/kconvl33samepool22max/kconvl33samepool22max_th11.cuh"

//======================= Use_t Forward ===========================

void kconvl33samepool22max_use_call_mode_th11(Use_t * use, uint inst, uint time) {
	Mdl_t * mdl = use->mdl;
	uint Ax=mdl->param[inst][0];
	uint Ay=mdl->param[inst][1];
	uint n0=mdl->param[inst][2];
	uint n1=mdl->param[inst][3];
	uint activ=mdl->param[inst][4];
	uint input_start=mdl->param[inst][5];
	uint ystart=mdl->param[inst][6];
	uint wstart=mdl->param[inst][7];

	//Copy Kernels to Constant memory
	kconvl33samepool22max_use_const_MemCpyToSymbol(use->weight + wstart, n0*n1*9);

	//	Kconvl with 'boundared' input image
	kconvl33samepool22max_use_const_th1x1<<<dim3(KERN_DIV(Ax/2,16), KERN_DIV(Ay/2,16), n1), dim3(16,16,1)>>>(
		n0, n1, Ax, Ay, 
		activ,
		time,
		mdl->total, mdl->weights,
		istart, wstart, ystart,
		use->var, use->weight);
};

//========================		Train_t	  =========================

//----------------------------- forward ---------------------------

void kconvl33samepool22max_forward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint Ax=mdl->param[inst][0],			\
		 Ay=mdl->param[inst][1],			\
		 n0=mdl->param[inst][2],			\
		 n1=mdl->param[inst][3],			\
		 activ=mdl->param[inst][4],		\
		 input_start=mdl->param[inst][5],	\
		 ystart=mdl->param[inst][6],		\
		 wstart=mdl->param[inst][7],		\
		 locdstart=mdl->param[inst][8],	\
		 drop_rate_int=mdl->param[inst][9];

	float drop_rate = drop_rate_int / 100;

	uint seed;

	for (uint set=0; set < train->sets; set++) {
		seed = (uint)pseudo_randomi(start_seed + set*inst);

		kconvl33samepool22max_train_const_MemCpyToSymbol(train->_weight + set*(mdl->weights)+wstart, n0*n1*9);
		
		kconvl33samepool22max_forward_const_th1x1<<<dim3(KERN_DIV(Ax/2,16), KERN_DIV(Ay/2,16), n1), dim3(16,16,1)>>>(
			n0, n1, Ax, Ay,
			activ,
			time,
			mdl->total, mdl->weights, mdl->locds,
			istart, wstart, ystart, locdstart,
			seed, drop_rate,
			set, train->sets,
			train->_var, train->_weight, train->_locd);

		hipDeviceSynchronize();
	}
};

//----------------------------- backward ---------------------------

void kconvl33samepool22max_backward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint Ax=mdl->param[inst][0],			\
		 Ay=mdl->param[inst][1],			\
		 n0=mdl->param[inst][2],			\
		 n1=mdl->param[inst][3],			\
		 activ=mdl->param[inst][4],		\
		 istart=mdl->param[inst][5],	\
		 ystart=mdl->param[inst][6],		\
		 wstart=mdl->param[inst][7],		\
		 locdstart=mdl->param[inst][8],	\
		 drop_rate_int=mdl->param[inst][9];

	float drop_rate = drop_rate_int / 100;

	uint seed;

	//
	for (uint set=0; set < train->sets; set++) {
		seed = (uint)pseudo_randomi(start_seed + set*inst);

		kconvl33samepool22max_train_const_MemCpyToSymbol(train->_weight + set*(mdl->weights)+wstart, n0*n1*9);
		
		kconvl33samepool22max_backward_const_th1x1<<<dim3(KERN_DIV(Ax/2,16), KERN_DIV(Ay/2,16), n1), dim3(16,16,1)>>>(
			n0, n1, Ax, Ay,
			activ,
			time,
			mdl->total, mdl->weights, mdl->locds,
			istart, wstart, ystart, locdstart,
			train->_var, train->_weight, train->_locd,
			train->_grad, train->_meand,
			seed, drop_rate,
			set, train->sets);

		hipDeviceSynchronize();
	}
};