#include "hip/hip_runtime.h"
#include "pkg_head/insts/gaussfiltre2d/gaussfiltre2d_th11.cuh"

__global__
void gaussfiltre2d_backward_th1x1(
	uint X, uint Y,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	float dlds;

	if (x < len) {
		dlds = grad[time*sets*total + set*total + ystart + (y*X+x)] * locd[time*sets*lsize + set*lsize + lstart + (y*X+x)];

		grad[time*sets*total + istart + (y*X+x)] += dlds;
		atomicAdd(meand + ws*set + wstart + x, dlds);
	}
};