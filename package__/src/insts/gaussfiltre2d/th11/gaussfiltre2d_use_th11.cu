#include "hip/hip_runtime.h"
#include "pkg_head/insts/gaussfiltre2d/gaussfiltre2d_th11.cuh"

__global__
void gaussfiltre1d_use_th1x1(
	uint X, uint Y,
	uint time,
	uint total,
	uint istart, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < X && y < Y) {
		var[time*total + ystart + y*X + x] = exp(-pow(var[time*total + istart + y*X + x] + weight[wstart + x],2));
	}
};