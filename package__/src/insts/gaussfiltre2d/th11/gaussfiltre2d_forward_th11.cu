#include "hip/hip_runtime.h"
#include "pkg_head/insts/gaussfiltre2d/gaussfiltre2d_th11.cuh"

__global__
void gaussfiltre2d_forward_th1x1(
	uint X, uint Y,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd,
	uint seed,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	float _tmp;

	if (x < X && y < Y) {
		_tmp = var[time*sets*total + istart + (y*X+x)] + weight[ws*set + wstart + x];
		var[time*sets*total + set*total + ystart + (y*X+x)] = exp(-pow(_tmp,2));
		locd[time*sets*lsize + set*lsize + lstart + (y*X+x)] = -2*_tmp*exp(-pow(_tmp,2));
	}
};