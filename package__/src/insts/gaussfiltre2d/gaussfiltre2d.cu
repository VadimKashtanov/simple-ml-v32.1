#include "hip/hip_runtime.h"
#include "pkg_head/insts/gaussfiltre2d/gaussfiltre2d.cuh"

//			   0  1    2      3      4       5
//Arguments = [X,Y, istart,ystart,wstart,lstart]

void gaussfiltre2d_check(uint * param) {
	if (param[0] == 0) raise(SIGINT);
	if (param[1] == 0) raise(SIGINT);
};

void gaussfiltre2d_cpu(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint X=mdl->param[inst][0],		\
		 Y=mdl->param[inst][1],		\
		 istart=mdl->param[inst][2],\
		 ystart=mdl->param[inst][3],\
		 wstart=mdl->param[inst][4];

	uint inp = total*time + istart;
	uint out = total*time + ystart;

	uint total = mdl->total;

	float * var = cpu->var;
	float * weight = mdl->weight;

	float a,p;

	for (uint y=0; y < Y; y++) {
		for (uint x=0; x < X; x++) {
			a = var[inp + (y*X+x)];
			p = weight[wstart + (x)];
			var[out + y*X + x] = exp(-(a+p)**2);
		}
	}
};

void gaussfiltre2d_use(Use_t * use, uint inst, uint time) {
	gaussfiltre2d_use_call_mode_th11(use, inst, time);
};

void gaussfiltre2d_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	gaussfiltre2d_forward_call_mode_th11(train, inst, time, start_seed);
};

void gaussfiltre2d_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	gaussfiltre2d_backward_call_mode_th11(train, inst, time, start_seed);
};