#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot2drecurent/dot2drecurent_th11.cuh"

__global__
void dot2drecurent_forward_th1x1(
	uint Ax, uint Ay, uint At, uint Bx,
	uint activ,
	uint time,
	uint istart, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd,
	uint seed, float drop_rate,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	float _tmp, _locd;

	uint Apos, Wpos;

	if (y < Ay && x < Bx) {
		_tmp = 0;

		Apos = (time-At)*total*sets + set*total + istart + y*Ax;
		Wpos = ws*set + wstart + y;

		for (uint i=0; i < Ax; i++) {
			if (pseudo_randomf(Apos*seed) >= drop_rate)
				_tmp += var[Apos] * weight[Wpos];
			Apos++;
			Wpos += Bx;
		}

		_tmp += weight[ws*set + wstart + Bx*Ax + (y*Bx + x)];	//==wstart + Ax*Yx + y      car on a deja +y, et on a += Ax*Yx (for i<Ax) {+=Yx}

		if (activ == 0)	{
			_tmp = 1 / (1 + exp(-_tmp));
			_locd = _tmp * (1 - _tmp);

		} else if (activ == 1) {
			_tmp = tanh(_tmp);
			_locd = 1 - _tmp*_tmp;

		} else if (activ == 2) {
			_locd = -2*_tmp;
			_tmp = exp(-_tmp*_tmp);
			_locd = _tmp * _locd;
		} else {
			_locd = (tmp > 0);
			_tmp = _tmp * _locd;
		}

		var[time*sets*total + set*total + ystart + (y*Bx + x)] = _tmp;
		locd[time*locdsize*sets + set*locdsize + locdstart + (y*Bx + x)] = _locd;
	}
}