#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot2drecurent/dot2drecurent_th11.cuh"

__global__
void dot2drecurent_use_th1x1(
	uint Ax, uint Ay, uint At, uint Bx,
	uint activ,
	uint time,
	uint total,
	uint istart, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	float _tmp;

	uint Apos, Wpos;

	if (y < Ay && x < Bx) {

		_tmp = 0;

		Apos = (time-At)*total + istart + y*Ax;
		Wpos = wstart + y;

		for (uint i=0; i < Ax; i++) {
			_tmp += var[Apos] * weight[Wpos];
			Apos++;
			Wpos += Bx;
		}

		_tmp += weight[wstart + Bx*Ax + (y*Bx + x)];	//==wstart + Ax*Yx + y      car on a deja +y, et on a += Ax*Yx (for i<Ax) {+=Yx}

		if (activ == 0)	_tmp = 1 / (1 + exp(-_tmp));
		else if (activ == 1) _tmp = tanh(_tmp);
		else if (activ == 2) _tmp = exp(-_tmp*_tmp);
		else _tmp *= (tmp > 0);

		var[time*total + ystart + (y*Bx + x)] = _tmp;
	}
};