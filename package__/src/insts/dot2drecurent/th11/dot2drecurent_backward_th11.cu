#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot2drecurent/dot2drecurent_th11.cuh"

__global__
void dot2drecurent_backward_th1x1(
	uint Ax, uint Ay, uint At, uint Bx,
	uint activ,
	uint time,
	uint istart, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.y;

	float dlds;
	uint Apos, Wpos;

	if (y < Yx) {
		dlds = grad[time*sets*total + set*total + ystart + (y*Bx + x)] * locd[time*locdsize*sets + set*locdsize + locdstart + (y*Bx + x)];

		Apos = (time-At)*total*sets + set*total + istart + y*Ax;
		Wpos = ws*set + wstart + y;

		for (uint i=0; i < Ax; i++) {
			if (pseudo_randomf(Apos*seed) >= drop_rate) {
				atomicAdd(grad + Apos, weight[Wpos] * dlds);
				atomicAdd(meand + Wpos, var[Apos] * dlds);
			}

			Apos++;
			Wpos += Bx;
		}

		meand[ws*set + wstart + Bx*Ax + (y*Bx + x)] += dlds;
	}
}