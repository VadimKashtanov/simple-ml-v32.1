#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot2drecurent/dot2drecurent.cuh"

//			   0  1    2   3    4      5       6    7			8
//Arguments = [Ax,Ay, At, Bx, istart,ystart,wstart,locdstart, drate]

void dot2drecurent_check(uint * param) {
	if (param[0] == 0) raise(SIGINT);
	if (param[1] == 0) raise(SIGINT);
	if (param[3] == 0) raise(SIGINT);
	if (param[8] >100) raise(SIGINT);
};

void dot2drecurent_cpu(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint Ax=mdl->param[inst][0],	\
		 Ay=mdl->param[inst][1],	\
		 At=mdl->param[inst][2],	\
		 Bx=mdl->param[inst][3],	\
		 istart=mdl->param[inst][4],\
		 ystart=mdl->param[inst][5],\
		 wstart=mdl->param[inst][6],\
		 lstart=mdl->param[inst][7],\
		 drate=mdl->param[inst][8];

	uint total = mdl->total;

	float * var = cpu->var;
	float * weight = mdl->weight;

	float tmp;

	uint Apos, Wpos;

	if (time - At >= 0) {
		for (uint y=0; y < Ay; y++) {
			for (uint x=0; x < Bx; x++) {
				_tmp = 0;

				Apos = (time-At)*total + istart + y*Ax;
				Wpos = wstart + y;

				for (uint i=0; i < Ax; i++) {
					_tmp += var[Apos] * weight[Wpos];
					Apos++;
					Wpos += Bx;
				}

				_tmp += weight[wstart + Ax*Bx + (y*Bx + x)];

				if (activ == 0)	_tmp = 1 / (1 + exp(-_tmp));
				else if (activ == 1) _tmp = tanh(_tmp);
				else if (activ == 2) _tmp = exp(-_tmp*_tmp);
				else _tmp *= (tmp > 0);

				var[time*total + ystart + (y*Bx + x)] = _tmp;
			}
		}
	}
};

void dot2drecurent_use(Use_t * use, uint inst, uint time) {
	dot2drecurent_use_call_mode_th11(use, inst, time);
};

void dot2drecurent_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	dot2drecurent_forward_call_mode_th11(train, inst, time, start_seed);
};

void dot2drecurent_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	dot2drecurent_backward_call_mode_th11(train, inst, time, start_seed);
};