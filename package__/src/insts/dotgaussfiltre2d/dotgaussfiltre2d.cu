#include "hip/hip_runtime.h"
#include "pkg_head/insts/dotgaussfiltre2d/dotgaussfiltre2d.cuh"

//			   0  1  2     3      4       5       6        7 
//Arguments = [Ax,Ay,Bx, istart,ystart,wstart,locdstart, drate]

/*

	  	      [p0,p1]
		      [p2,p3]
		      [p4,p5]
[a0,a1,a2] -> [y0,y1]
[a3,a4,a5] -> [y2,y3]

Y[y*Bx + x] = sum( exp(-(a[y*Ax + i] + p[i*Bx + x])^2) for i in range(Ax))

y0 = exp(-(a0+p0)^2) + exp(-(a1+p2)^2) + exp(-(a2+p4)^2)
y3 = exp(-(a3+p1)^2) + exp(-(a4+p3)^2) + exp(-(a5+p5)^2)

locd = -2(a+p)y

*/

void dotgaussfiltre2d_check(uint * param) {
	if (param[0] == 0) raise(SIGINT);
	if (param[1] == 0) raise(SIGINT);
	if (param[2] == 0) raise(SIGINT);
	if (param[7] >100) raise(SIGINT);
};

void dotgaussfiltre2d_cpu(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint Ax=mdl->param[inst][0],		\
		 Ay=mdl->param[inst][1],		\
		 Bx=mdl->param[inst][2],		\
		 istart=mdl->param[inst][3],	\
		 ystart=mdl->param[inst][4],	\
		 wstart=mdl->param[inst][5],	\
		 lstart=mdl->param[inst][6],	\
		 drate=mdl->param[inst][7];

	uint inp = total*time + istart;
	uint W = wstart;
	uint out = total*time + ystart;

	uint total = mdl->total;

	float * var = cpu->var;
	float * weight = mdl->weight;

	float _tmp;

	for (uint y=0; y < Ay; y++) {
		for (uint x=0; x < Bx; x++) {
			_tmp = 0;

			for (uint i=0; i < Ax; i++) {

				apos = time*total + istart + y*Ax + i;
				ppos = wstart + i*Bx + x;
				
				_tmp += exp(-pow(var[apos] + weight[ppos],2));
			}

			var[time*total + ystart + (y*Bx+x)] = _tmp;
		}
	}
};

void dotgaussfiltre2d_use(Use_t * use, uint inst, uint time) {
	dotgaussfiltre2d_use_call_mode_th11(use, inst, time);
};

void dotgaussfiltre2d_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	dotgaussfiltre2d_forward_call_mode_th11(train, inst, time, start_seed);
};

void dotgaussfiltre2d_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	dotgaussfiltre2d_backward_call_mode_th11(train, inst, time, start_seed);
};