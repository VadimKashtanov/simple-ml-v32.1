#include "hip/hip_runtime.h"
#include "pkg_head/insts/dotgaussfiltre2d/dotgaussfiltre2d_th11.cuh"

__global__
void dotgaussfiltre2d_use_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint activ,
	uint time,
	uint total,
	uint istart, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockIdx.y;

	float _tmp;

	if (x < Bx && y < Ay) {

		for (uint y=0; y < Ay; y++) {
			for (uint x=0; x < Bx; x++) {
				_tmp = 0;

				for (uint i=0; i < Ax; i++) {

					apos = time*total + istart + y*Ax + i;
					ppos = wstart + i*Bx + x;
					
					_tmp += exp(-pow(var[apos] + weight[ppos],2));
				}

				var[time*total + ystart + (y*Bx+x)] = _tmp;
			}
		}

	}
};