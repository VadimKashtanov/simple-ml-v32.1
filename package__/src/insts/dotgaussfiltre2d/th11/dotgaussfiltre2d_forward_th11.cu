#include "hip/hip_runtime.h"
#include "pkg_head/insts/dotgaussfiltre2d/dotgaussfiltre2d_th11.cuh"

__global__
void dotgaussfiltre2d_forward_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint activ,
	uint time,
	uint istart, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd,
	uint seed, float drop_rate,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockIdx.y;
	uint set = blockIdx.z;

	float _sum, _tmp;

	if (x < Bx && y < Ay) {

		_sum = 0;

		for (uint i=0; i < Ax; i++) {
			apos = time*sets*total + set*total + istart + y*Ax + i;

			if (pseudo_randomf(apos*seed) >= drop_rate) {
				ppos = ws*set + wstart + i*Bx + x;

				_tmp = var[apos] + weight[ppos];
						
				_sum += exp(-pow(_tmp, 2));

				locd[time*sets*lsize + set*lsize + lstart + Ax*(y*Bx+x) + i] = -2*(_tmp)*exp(-(_tmp)**2);
			} /*else {
			var[apos] == 0;
			}*/
		}
		var[time*sets*total + set*total + ystart + (y*Bx+x)] = _sum;
	}
};