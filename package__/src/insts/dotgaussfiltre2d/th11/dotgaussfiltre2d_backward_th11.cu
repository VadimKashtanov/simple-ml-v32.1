#include "hip/hip_runtime.h"
#include "pkg_head/insts/dotgaussfiltre2d/dotgaussfiltre2d_th11.cuh"

__global__
void dotgaussfiltre2d_backward_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint activ,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	float _grad, dlds;

	if (x < Bx && y < Ay) {

		_grad = grad[time*sets*total + set*total + ystart + (y*Bx+x)];

		for (uint i=0; i < Ax; i++) {
			apos = time*sets*total + set*total + istart + y*Ax + i;

			if (pseudo_randomf(apos*seed) >= drop_rate) {
				ppos = ws*set + wstart + i*Bx + x;

				dlds = locd[time*sets*lsize + set*lsize + lstart + Ax*(y*Bx+x) + i] * _grad;

				atomicAdd(grad + apos, dlds);
				atomicAdd(meand + ppos, dlds);
			}
		}
	}
}
	/*uint x = threadIdx.x + blockIdx.x * blockDim.x;		//	Ax
	uint y = threadIdx.y + blockIdx.y * blockDim.y;		//	Ay
	uint set = blockIdx.z;

	uint apos = time*sets*total + set*total + istart + y*Ax + x;

	if (x < Ax && y < Ay && pseudo_randomf(apos)) {
		float input_value = var[apos];
		float __grad_input = 0;
		float wpos;

		for (uint i=0; i < Bx; i++) {
			wpos = ws*set + wstart + x*Bx + x;

			grad[ws*set + wstart + ] += input_value * 
			__grad_input += weight[] * ;
		};
	};
};