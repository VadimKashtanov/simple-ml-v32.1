#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot2d/dot2d.cuh"

void dot2d_check(uint * param) {
	if (param[0] == 0) raise(SIGINT);
	if (param[1] == 0) raise(SIGINT);
	if (param[2] == 0) raise(SIGINT);
	if (param[8] >100) raise(SIGINT);
	if (param[3] >= 4) raise(SIGINT);
};

void dot2d_cpu_call(
	Cpu_t * cpu, uint inst, uint time)
{
	Mdl_t * mdl = cpu->mdl;

	uint Ax=mdl->param[inst][0],			\
		 Ay=mdl->param[inst][1],			\
		 Bx=mdl->param[inst][2],			\
		 activ=mdl->param[inst][3],			\
		 input_start=mdl->param[inst][4],	\
		 ystart=mdl->param[inst][5],		\
		 wstart=mdl->param[inst][6];

	float * var = cpu->var;
	float * weight = mdl->weight;

	float _tmp;
	uint _inp, _w;
	
	for (uint x=0; x < Bx; x++) {
		for (uint y=0; y < Ay; y++) {
			_tmp = 0;

			//Scalar product of 2 vectors in input (A) and weight (B)
			_inp = time*mdl->total + input_start + Ax*y;
			_w = wstart + x;
			for (uint i=0; i < Ax; i++) {
				_tmp += var[_inp + i] * weight[_w + i*Bx];
			}

			//Adding bias
			_tmp = _tmp + weight[wstart + Bx*Ax + y*Bx + x];

			//Activation
			if (activ == 0) _tmp = 1 / (1 + exp(-_tmp));
			else if (activ == 1) _tmp = tanh(_tmp);
			else if (activ == 2) _tmp = exp(-_tmp*_tmp);
			else _tmp = _tmp * (_tmp >= 0);

			//Write it to Y
			var[time*mdl->total + ystart + y*Bx + x] = _tmp;
		}
	}
};

void dot2d_use_call(Use_t * use, uint inst, uint time) {
	dot2d_use_call_mode_th11(use, inst, time);
};

void dot2d_forward_call(Train_t * train, uint inst, uint time, uint start_seed) {
	dot2d_forward_call_mode_th11(train, inst, time, start_seed);
};

void dot2d_backward_call(Train_t * train, uint inst, uint time, uint start_seed) {
	dot2d_backward_call_mode_th11(train, inst, time, start_seed);
};