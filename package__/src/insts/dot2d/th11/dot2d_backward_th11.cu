#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot2d.cuh"

__global__
void dot2d_backward_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint activ,
	uint time,
	uint input_start, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	uint Yx = threadIdx.x + blockIdx.x*blockDim.x,	\
		 Yy = threadIdx.y + blockIdx.y*blockDim.y,	\
		 set = threadIdx.z + blockIdx.z*blockDim.z;

	uint Apos = time*sets*total + set*total + input_start + Yy*Ax;
	uint weight_start = set*wsize + wstart;
	uint Bpos = weight_start + Yx;

	uint Y_pos = Yy*Bx + Yx;

	float dlds = grad[time*sets*total + set*total + ystart + Y_pos] * locd[time*sets*locdsize + set*locdsize + locdstart + Y_pos];

	meand[weight_start + Bx*Ax + Yy*Bx + Yx] += dlds;

	for (uint i=0; i < Ax; i++) {
		if (pseudo_randomf(Apos + seed) >= drop_rate) {
			atomicAdd(&grad[Apos], dlds * weight[Bpos]);
			atomicAdd(&meand[Bpos], dlds * var[Apos]);
		}
		Apos++;
		Bpos += Bx;
	}
};

//=============================================================================================

__global__
void dot2d_backward_th1x1_bias(
	uint Ax, uint Ay, uint Bx,
	uint activ,
	uint time,
	uint input_start, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	/*	Kernel coordinates	*/
	uint _Yx = blockIdx.x + blockIdx.x*blockDim.x,	\
		 _Yy = blockIdx.y + blockIdx.y*blockDim.y,	\
		 set = blockIdx.z + blockIdx.z*blockDim.z;

	/*	Train_t starts */
	uint time_sets = time*sets + set;
	uint y_pos = _Yy*Bx + _Yx;
	uint trt_out = time_sets*total + ystart + y_pos;
	uint trt_locd = time_sets*locdsize + locdstart + y_pos;

	meand[set*wsize + wstart + Ax*Bx + _Yy*Bx + _Yx] += locd[trt_locd] * grad[trt_out];
};

__global__
void dot2d_backward_th1x1_input(		//grad input = dLdS @ W.T
	uint Ax, uint Ay, uint Bx,
	uint activ,
	uint time,
	uint input_start, uint ystart, uint wstart, uint locdstart,	//in a mdl line
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	/*	Kernel coordinates	*/
	uint _Ax = blockIdx.x + blockIdx.x*blockDim.x,	\
		 _Ay = blockIdx.y + blockIdx.y*blockDim.y,	\
		 set = blockIdx.z + blockIdx.z*blockDim.z;

	/*	Train_t starts */
	uint time_sets = time*sets + set;
	uint trt_out = time_sets*total + ystart + _Ay*Bx;
	uint trt_locd = time_sets*locdsize + locdstart + _Ay*Bx;
	uint trt_A = time_sets*total + input_start + _Ay*Ax + _Ax;
	uint trt_B = set*wsize + wstart + _Ax*Bx;

	if (pseudo_randomf(trt_A + seed) >= drop_rate)
	{
		float tmp = 0;

		for (uint i=0; i < Bx; i++)
			tmp += weight[trt_B + i] * (locd[trt_locd + i] * grad[trt_out + i]);

		grad[trt_A] += tmp;
	}
}

__global__
void dot2d_backward_th1x1_weight(		//grad weight = input.T @ dLdS
	uint Ax, uint Ay, uint Bx,
	uint activ,
	uint time,
	uint input_start, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	/*	Kernel coordinates	*/
	uint _Bx = blockIdx.x + blockIdx.x*blockDim.x,	\
		 _By = blockIdx.y + blockIdx.y*blockDim.y,	\
		 set = blockIdx.z + blockIdx.z*blockDim.z;

	/*	Train_t starts */
	uint time_sets = time*sets + set;
	uint trt_out = time_sets*total + ystart + _Bx;
	uint trt_locd = time_sets*locdsize + locdstart + _Bx;
	uint trt_A = time_sets*total + input_start + _By;
	uint trt_B = set*wsize + wstart + _By*Bx + _Bx;

	float tmp = 0;

	for (uint i=0; i < Ay; i++) {
		if (pseudo_randomf((trt_A + i*Ax) + seed) >= drop_rate)
			tmp += var[trt_A + i*Ax] * (locd[trt_locd + i*Bx] * grad[trt_out + i*Bx]);
	}

	meand[trt_B] += tmp;
};