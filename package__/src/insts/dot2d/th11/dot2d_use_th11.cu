#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot2d.cuh"

__global__
void dot2d_use_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint activ,
	uint time,
	uint vars,
	uint input_start, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint Yx = threadIdx.x + blockIdx.x*blockDim.x,	\
		 Yy = threadIdx.y + blockIdx.y*blockDim.y;	\

	uint Apos = time*vars + input_start + Yy*Ax;
	uint Bpos = wstart + Yx;

	float sum = 0;
	for (uint i=0; i < Ax; i++) {
		sum += var[Apos] * weight[Bpos];
		Apos++;
		Bpos += Bx; 
	}
	sum += weight[wstart + Bx*Ax + Yy*Bx + Yx];

	if (activ == 0) sum = 1 / (1 + exp(-sum));
	else if (activ == 1) sum = tanh(sum);
	else if (activ == 2) sum = exp(-pow(sum,2));
	else sum = sum*(sum >= 0);

	var[time*vars + ystart + Yy*Bx + Yx] = sum;
};