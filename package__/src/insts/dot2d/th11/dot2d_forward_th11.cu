#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot2d.cuh"

__global__
void dot2d_forward_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint activ,
	uint time,
	uint input_start, uint ystart, uint wstart, uint locdstart,
	uint linesize, uint wsize, uint locdsize,
	float * var, float * weight, float * locd,
	uint seed, float drop_rate,
	uint sets)
{
	uint Yx = threadIdx.x + blockIdx.x*blockDim.x,	\
		 Yy = threadIdx.y + blockIdx.y*blockDim.y,	\
		 set = threadIdx.z + blockIdx.z*blockDim.z;

	uint Apos = time*sets*linesize + set*linesize + input_start + Yy*Ax;
	uint weight_start = set*wsize + wstart;
	uint Bpos = weight_start + Yx;

	float sum = 0;
	for (uint i=0; i < Ax; i++) {
		if (pseudo_randomf(Apos + seed) >= drop_rate)
			sum += var[Apos] * weight[Bpos];
		Apos++;
		Bpos += Bx;
	}
	sum += weight[weight_start + Bx*Ax + Yy*Bx + Yx];
	
	float __locd;

	if (activ == 0) {
		sum = 1 / (1 + exp(-sum));
		__locd = sum*(1 - sum);	//f'(x) = f(x)(1 - f(x))
	} else if (activ == 1) {
		sum = tanh(sum);
		__locd = 1 - sum*sum;	//f'(x) = 1 - tanh(x)^2
	} else if (activ == 2) {
		__locd = sum;
		sum = exp(-pow(sum,2));
		__locd = -2*__locd*sum;	//f'(x) = -2x*e^(-x^2)
	} else {
		__locd = (sum >= 0);
		sum = sum*__locd;
	}

	var[time*sets*linesize + set*linesize + ystart + Yy*Bx + Yx] = sum;		//same assembler than putting it in if/else structure
	locd[time*sets*locdsize + set*locdsize + locdstart + Yy*Bx + Yx] = __locd;
};