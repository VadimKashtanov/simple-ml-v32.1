#include "hip/hip_runtime.h"
#include "pkg_head/insts/lstm1d.cuh"

__global__
void lstm1d_forward_th1x1(
	uint X, uint Y,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd,
	uint seed, float drop_rate,
	uint sets)
{
	//	<<<Y,sets>>>
	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	if (y < Y) {
		uint inp = total*sets*time + total*set + istart;
		uint W = ws*set + wstart;
		uint out = total*sets*time + total*set + ystart;
		uint locdpos = lsize*sets*time + lsize*set + lstart;

		uint _W = X*Y;
		uint _U = Y*Y;
		uint _B = Y;

		uint lineW = _W + _U + _B;

		uint vpos, wpos;

		// f0,f1,f2 = logistic(x@W + h[-1]@U + B)
		// g0 	  = tanh 	(x@W + h[-1]@U + B)
		float f0=0,f1=0,f2=0,g0=0;

		float tmpt;

		// .W
		for (uint k=0; k < X; k++) {	//for all in INPUT
			//	Positions
			vpos = inp + k;

			//	Drop
			if (pseudo_randomf(seed + vpos) > drop_rate) {	//pas de drop sur h[-1]@.U car h[-1] n'est pas un input mais un output et h[-1] aura deja l'influence
				
				//
				wpos = k*Y + x;

				tmpt = var[vpos];
				f0 += tmpt * weight[W + 0*lineW + wpos];
				f1 += tmpt * weight[W + 1*lineW + wpos];
				f2 += tmpt * weight[W + 2*lineW + wpos];
				g0 += tmpt * weight[W + 3*lineW + wpos];
			}
		}

		// .U
		if (time > 0) {
			for (uint k=0; k < Y; k++) {
				vpos = total*sets*(total-1) + total*set + ystart + Y + k;
				wpos = _W + k*Y + x;

				tmpt = var[vpos];
				f0 += tmpt * weight[W + 0*lineW + wpos];
				f1 += tmpt * weight[W + 1*lineW + wpos];
				f2 += tmpt * weight[W + 2*lineW + wpos];
				g0 += tmpt * weight[W + 3*lineW + wpos];
			}
		}

		// .B
		wpos = _W + _U + y;
		f0 += w[W + 0*lineW + wpos];
		f1 += w[W + 1*lineW + wpos];
		f2 += w[W + 2*lineW + wpos];
		g0 += w[W + 3*lineW + wpos];

		// activ(_sum)
		f0 = logistic(f0);
		f1 = logistic(f1);
		f2 = logistic(f2);
		g0 = tanh(g0);

		// e = f0 * e[-1] + f1 * g0
		// l - 1 have to be >= 0
		float e_1;
		if (time > 0) e_1 = var[total*sets*(time-1) + total*set + ystart + y];
		else e_t = 0;
		
		float e = f0*e_1 + f1*g0;
		float h = f2 * e;

		locd[locdpos + 0*Y + y] = f0;//f2*e_1*( f0*(1 - f0) );	//	f0 locd
		locd[locdpos + 1*Y + y] = f1;//f2*g0*( f1*(1 - f1) );	//	f1 locd
		locd[locdpos + 2*Y + y] = f2;//e*( f2*(1 - f2) );		//	f2 locd
		lcod[locdpos + 3*Y + y] = g0;//f2*f1*( 1 - g0*g0);		//	g0 locd

		var[out + 0*Y + y] = e;
		var[out + 1*Y + y] = h;
	}
};