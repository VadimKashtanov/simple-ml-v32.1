#include "hip/hip_runtime.h"
#include "pkg_head/insts/lstm1d.cuh"

/*			  *=====*
			  |     |
			  |		|
			  |	.W	|
			  |		|
			  |		|
			  *=====*
*===========* *=====*
|	.input	| | 	|	input@W
*===========* *=====*
				 +
			  *=====*
			  |	.U  |
			  |	    |
			  *=====*
	*=======* *=====*
	| h[-1] | |		|  h[-1]@U
	*=======* *=====*
				 +
			  *=====*
			  |	.B	|
			  *=====*
*/

__global__
void lstm1d_backward_INPUT_th1x1(
	uint X, uint Y,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drate,
	uint sets)
{
	/*	Backward grad(input)
		meand(.W) of f0,f1,f2,g0
	*/

	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	uint inp = total*sets*time + total*_set + istart + x;

	if (x < X && pseudo_randomf(seed + inp) > drop_rate) {	//if input[x] is droped, following will be *0

		//
		//	each `x` use all the locd, so an dynamic shared memory could be usefull
		//	save on extern __shared__ dsf0,dsf1,dsf2,dsg0

		float grad_input_compute = 0;	//_INPUT_ et _H1_ ajoutent un gradient a input[x]

		uint W = ws*set + wstart;
		uint out = total*sets*time + total*set + ystart;
		uint locdpos = lsize*sets*time + lsize*set + lstart;

		uint lineW = X*Y + Y*Y + Y;

		uint vpos = total*sets*time + set*total + istart + x;
		float xval = var[vpos];

		float chain_deriv;
		float dH,f0,f1,f2,g0,de;
		float dsf0, dsf1, dsf2, dsg0;

		uint wpos, epos, e_1pos, hpos;

		//	Backward W
		for (uint k=0; k < Y; k++) {	//[ w0 w1 w2 w3 ... wn]	une ligne du .W (la premiere par exemple)
										//car inp[x] est multiplice par `w[x*Bx + k] for k in Bx`  ou Bx==Y

			epos = out + k;
			e_1pos = total*sets*(time-1) + total*_set + ystart + k; //if l == 0 , e_1pos <= 0
			hpos = out + Y + k;

			dH = grad[hpos];

			f0 = locd[locdpos + 0*Y + k];// * dH;
			f1 = locd[locdpos + 1*Y + k];// * dH;
			f2 = locd[locdpos + 2*Y + k];// * dH;
			g0 = lcod[locdpos + 3*Y + k];// * dH;

			de = grad[epos] + dH * f2;	//grad(e) += dH*f2

			grad[epos] = de;

			//if time > 0:
			grad[e_1pos] += de * f0;

			dsf0 = de * var[e_1pos] * f0 * (1 - f0);
			dsf1 = de * g0 * f1 * (1 - f1);
			dsf2 = dH * e * f2 * (1 - f2);
			dsg0 = de * f1 * (1 - g0*g0);

			//	f0
			wpos = ws*set + wstart + 0*lineW + (x*Y + k);					//on met a jour que .W pas .U no .B
			//meand[wpos] += dsf0 * xvalue;
			atomicAdd(meand + wpos, dsf0 * xvalue);
			grad_input_compute += dsf0 * weight[wpos];

			//	f1
			wpos = ws*set + wstart + 1*lineW + (x*Y + k);			//on met a jour que .W pas .U no .B
			//meand[wpos] += dsf1 * xvalue;
			atomicAdd(meand + wpos, dsf1 * xvalue);
			grad_input_compute += dsf1 * weight[wpos];

			//	f2
			wpos = ws*set + wstart + 2*lineW + (x*Y + k);			//on met a jour que .W pas .U no .B
			//meand[wpos] += dsf2 * xvalue;
			atomicAdd(meand + wpos, dsf2 * xvalue);
			grad_input_compute += dsf2 * weight[wpos];

			//	g0
			wpos = ws*set + wstart + 3*lineW + (x*Y + k);			//on met a jour que .W pas .U no .B
			//meand[wpos] += dsf2 * xvalue;
			atomicAdd(meand + wpos, dsg0 * xvalue);
			grad_input_compute += dsf2 * weight[wpos];
		}

		//	Backward input
		grad[vpos] += grad_input_compute;
		//atomicAdd(grad + vpos, grad_input_compute);
	}
}

__global__
void lstm1d_backward_H1_BIAS_th1x1(
	uint X, uint Y,
	uint time,
	uint input_start, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsizeize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	//
	//	h[-1] @ .U
	//

	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	if (y < Y) {	//Only input is under drop. h is an output. It's values, could be droped, but in an other instruction

		float grad_H1_compute = 0;	//_INPUT_ et _H1_ ajoutent un gradient a input[x]

		//time > 0 because lstm1d.cu call this kernel only on time > 0
		uint h1pos = total*sets*(time-1) + set*total + istart + Y + y;
		float h1val = var[h1pos];

		uint W = ws*set + wstart;
		uint out = total*sets*time + total*set + ystart;
		uint locdpos = lsize*sets*time + lsize*set + lstart;

		uint _W = X*Y;
		uint _U = Y*Y;
		uint _B = Y;

		uint lineW = _W + _U + _B;
		
		float chain_deriv;
		float dH,f0,f1,f2,g0,de;
		float dsf0, dsf1, dsf2, dsg0;

		uint wpos, epos, e_1pos, hpos;
		for (uint k=0; k < Y; k++) {	//[ w0 w1 w2 w3 ... wn]	une ligne du .W (la premiere par exemple)
										//car inp[x] est multiplice par `w[x*Bx + k] for k in Bx`  ou Bx==Y
			epos = out + k;
			e_1pos = total*sets*(time-1) + total*_set + ystart + k; //if l == 0 , e_1pos <= 0
			hpos = out + Y + k;

			dH = grad[hpos];

			f0 = locd[locdpos + 0*Y + k];// * dH;
			f1 = locd[locdpos + 1*Y + k];// * dH;
			f2 = locd[locdpos + 2*Y + k];//* dH;
			g0 = lcod[locdpos + 3*Y + k];// * dH;

			de = grad[epos] + dH * f2;	//grad(e) += dH*f2

			grad[epos] = de;

			//if time > 0:
			grad[e_1pos] += de * f0;

			dsf0 = de * var[e_1pos] * f0 * (1 - f0);
			dsf1 = de * g0 * f1 * (1 - f1);
			dsf2 = dH * e * f2 * (1 - f2);
			dsg0 = de * f1 * (1 - g0*g0);

			//	f0
			wpos = ws*set + wstart + 0*lineW + _W + (x*Y + k);					//on met a jour que .U pas .W no .B
			//meand[wpos] += dsf0 * h1val;
			atomicAdd(meand + wpos, dsf0 * h1val);
			grad_H1_compute += dsf0 * weight[wpos];

			//	f1
			wpos = ws*set + wstart + 1*lineW + _W + (x*Y + k);			//on met a jour que .U pas .W no .B
			//meand[wpos] += dsf1 * h1val;
			atomicAdd(meand + wpos, dsf1 * h1val);
			grad_H1_compute += dsf1 * weight[wpos];

			//	f2
			wpos = ws*set + wstart + 2*lineW + _W + (x*Y + k);			//on met a jour que .U pas .W no .B
			//meand[wpos] += dsf2 * h1val;
			atomicAdd(meand + wpos, dsf2 * h1val);
			grad_H1_compute += dsf2 * weight[wpos];

			//	g0
			wpos = ws*set + wstart + 3*lineW + _W + (x*Y + k);			//on met a jour que .U pas .W no .B
			//meand[wpos] += dsg0 * h1val;
			atomicAdd(meand + wpos, dsg0 * h1val);
			grad_H1_compute += dsg0 * weight[wpos];
		}

		//	Backward h[-1]
		grad[h1pos] += grad_H1_compute;
		//atomicAdd(grad + h1pos, grad_H1_compute);

		//  ============================================
		//	Backward .B
		//	Vu que la grille est de <<<Y>>> on en profite car .B l'est aussi
		//	Au lieu de cree un autre fonction qui compute le gradient de .B, on le fait directe ici.	
		//
		
		epos = out + y;
		e_1pos = total*sets*(time-1) + total*_set + ystart + y; //if l == 0 , e_1pos <= 0
		hpos = out + Y + y;

		dH = grad[hpos];

		f0 = locd[locdpos + 0*Y + y];// * dH;
		f1 = locd[locdpos + 1*Y + y];// * dH;
		f2 = locd[locdpos + 2*Y + y];// * dH;
		g0 = lcod[locdpos + 3*Y + y];// * dH;

		de = grad[epos] + dH * f2;	//grad(e) += dH*f2
		grad[epos] = de;

		//if time > 0:
		grad[e_1pos] += de * f0;

		dsf0 = de * var[e_1pos] * f0 * (1 - f0);
		dsf1 = de * g0 * f1 * (1 - f1);
		dsf2 = dH * e * f2 * (1 - f2);
		dsg0 = de * f1 * (1 - g0*g0);

		//	f0
		meand[ws*set + wstart + 0*lineW + _W + _U + (x*Y + k)] += dsf0;

		//	f1
		meand[ws*set + wstart + 1*lineW + _W + _U + (x*Y + k)] += dsf1;

		//	f2
		meand[ws*set + wstart + 2*lineW + _W + _U + (x*Y + k)] += dsf2;

		//	g0
		meand[ws*set + wstart + 3*lineW + _W + _U + (x*Y + k)] += dsg0;
	}
};

__global__
void lstm1d_backward_BIAS_ONLY_th1x1(
	uint X, uint Y,
	uint time,
	uint input_start, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsizeize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	//
	//	h[-1] @ .U
	//

	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	if (y < Y) {	//Only input is under drop, .h is an output. It's values, could be droped, but in an other instruction

		uint W = ws*set + wstart;
		uint out = total*sets*time + total*set + ystart;
		uint locdpos = lsize*sets*time + lsize*set + lstart;

		uint _W = X*Y;
		uint _U = Y*Y;
		uint _B = Y;

		uint lineW = _W + _U + _B;

		float chain_deriv;
		float _grad;	//of h[t]
		float dH,f0,f1,f2,g0,de;
		float dsf0, dsf1, dsf2, dsg0;

		uint wpos, epos, e_1pos, hpos;

		//  ============================================
		//	Backward .B
		//	Vu que la grille est de <<<Y>>> on en profite car .B l'est aussi
		//	Au lieu de cree un autre fonction qui compute le gradient de .B, on le fait directe ici.	
		//
		
		epos = out + y;
		e_1pos = total*sets*(time-1) + total*_set + ystart + y; //if l == 0 , e_1pos <= 0
		hpos = out + Y + y;

		dH = grad[hpos];

		f0 = locd[locdpos + 0*Y + y];// * dH;
		f1 = locd[locdpos + 1*Y + y];// * dH;
		f2 = locd[locdpos + 2*Y + y];// * dH;
		g0 = lcod[locdpos + 3*Y + y];// * dH;

		de = grad[epos] + dH * f2;	//grad(e) += dH*f2
		grad[epos] = de;

		//if time > 0:
		grad[e_1pos] += de * f0;

		dsf0 = de * var[e_1pos] * f0 * (1 - f0);
		dsf1 = de * g0 * f1 * (1 - f1);
		dsf2 = dH * e * f2 * (1 - f2);
		dsg0 = de * f1 * (1 - g0*g0);

		//	f0
		meand[ws*set + wstart + 0*lineW + _W + _U + (x*Y + k)] += dsf0;

		//	f1
		meand[ws*set + wstart + 1*lineW + _W + _U + (x*Y + k)] += dsf1;

		//	f2
		meand[ws*set + wstart + 2*lineW + _W + _U + (x*Y + k)] += dsf2;

		//	g0
		meand[ws*set + wstart + 3*lineW + _W + _U + (x*Y + k)] += dsg0;
	}
};