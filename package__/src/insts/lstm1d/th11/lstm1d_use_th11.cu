#include "hip/hip_runtime.h"
#include "pkg_head/insts/lstm1d.cuh"

//			   0  1    2      3      4       5          6 
//Arguments = [X, Y, istart,ystart,wstart,locdstart, drop_rate]

__global__
void lstm1d_use_th1x1(
	uint X, uint Y,
	uint time,
	uint total,
	uint istart, uint ystart, uint wstart,
	float * var, float * weight)
{
	/*   
			<<< grid==dim3(Y)  block==dim3(1) >>>
	*/

	uint y = threadIdx.x + blockIdx.x * blockDim.x;

	if (y < Y) {

		uint inp = total*line + istart;
		uint W = wstart;

		uint lineW = X*Y + Y*Y + Y;	// == sizeof(W + U + B). There is 4 sets of (W,U,B) for f0,f1,f2 and g0

		uint vpos, wpos;

		float f0=0, f1=0, f2=0, g0=0;

		float tmpt;

		//	x @ W
		for (uint k=0; k < X; k++) {
			vpos = inp + k;
			wpos = k*X + y;

			tmpt = var[vpos];
			f0 += tmpt * weight[W + wpos];
			f1 += tmpt * weight[W + lineW + wpos];
			f2 += tmpt * weight[W + 2*lineW + wpos];
			g0 += tmpt * weight[W + 3*lineW + wpos];
		}

		//	h[-1] @ U
		if (time > 0) {
			for (uint k=0; k < X; k++) {
				vpos = (time-1)*total + istart + Y + k; 		//out - total == total*(l-1) + ystart
				wpos = X*Y + k*X + y;

				tmpt = var[vpos];
				f0 += tmpt * weight[W + wpos];
				f1 += tmpt * weight[W + lineW + wpos];
				f2 += tmpt * weight[W + 2*lineW + wpos];
				g0 += tmpt * weight[W + 3*lineW + wpos];
			}
		}

		//	+ B
		wpos = X*Y + Y*Y + y;
		f0 += weight[W + wpos];
		f1 += weight[W + lineW + wpos];
		f2 += weight[W + 2*lineW + wpos];
		g0 += weight[W + 3*lineW + wpos];

		f0 = logistic(f0);
		f1 = logistic(f1);
		f2 = logistic(f2);
		g0 = tanh(g0);

		if (time > 0) e_1 = var[(time-1)*total + ystart + y];
		else e_1 = 0; 
		
		float e = f0*e_1 + f1*g0;
		float h = f2 * e;

		var[time*total + ystart + y] = e;
		var[time*total + ystart + Y + y] = h;
	}
};