#include "hip/hip_runtime.h"
#include "pkg_head/insts/lstm1d/lstm1d.cuh"

//			   0  1    2      3      4       5          6 
//Arguments = [X, Y, istart,ystart,wstart,locdstart, drop_rate]

void lstm1d_check(uint * param) {
	if (param[0] == 0) raise(SIGINT);
	if (param[1] == 0) raise(SIGINT);
	if (param[6] >100) raise(SIGINT);
};

void lstm1d_cpu(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint X=mdl->param[inst][0],		\
		 Y=mdl->param[inst][1],		\
		 istart=mdl->param[inst][2],\
		 ystart=mdl->param[inst][3],\
		 wstart=mdl->param[inst][4];

	uint total = mdl->total;
	
	uint inp = total*time + istart;
	uint W = wstart;
	uint out = total*time + ystart;

	uint lineW = X*Y + Y*Y + Y;	// == sizeof(W + U + B). There is 4 sets of (W,U,B) for f0,f1,f2 and g0

	float * var = cpu->var;
	float * weight = mdl->weight;

	float vpos, wpos;

	float f0,f1,f2,g0,  e_1, e, h;

	for (uint y=0; y < Y; y++) {
		f0=0; f1=0; f2=0; g0=0;

		//	x @ W
		for (uint k=0; k < X; k++) {
			vpos = inp + k;
			wpos = k*X + y;
			f0 += var[vpos]*w[W + wpos];
			f1 += var[vpos]*w[W + lineW + wpos];
			f2 += var[vpos]*w[W + 2*lineW + wpos];
			g0 += var[vpos]*w[W + 3*lineW + wpos];
		}

		//	h[-1] @ U
		if (time > 0) {
			for (uint k=0; k < X; k++) {
				vpos = (time-1)*total + istart + Y + k; 		//out - total == total*(l-1) + ystart
				wpos = X*Y + k*X + y;
				f0 += var[vpos]*w[W + wpos];
				f1 += var[vpos]*w[W + lineW + wpos];
				f2 += var[vpos]*w[W + 2*lineW + wpos];
				g0 += var[vpos]*w[W + 3*lineW + wpos];
			}
		}

		//	+ B
		wpos = X*Y + Y*Y + y;
		f0 += w[W + wpos];
		f1 += w[W + lineW + wpos];
		f2 += w[W + 2*lineW + wpos];
		g0 += w[W + 3*lineW + wpos];

		// activate(_sum)
		f0 = logistic(f0);
		f1 = logistic(f1);
		f2 = logistic(f2);
		g0 = tanh(g0);

		// e = f0 * e[-1] + f1 * g0
		// l - 1 have to be >= 0 || l > 0
		if (l > 0) e_1 = var[total*(time-1) + ystart + y];
		else e_1 = 0;
		
		e = f0*e_1 + f1*g0;
		h = f2 * (e);	//f(x)=x

		var[out + y] = e;
		var[out + Y + y] = h;
	};
};

void lstm1d_use(Use_t * use, uint inst, uint time) {
	lstm1d_use_call_mode_th11(use, inst, time);
};

void lstm1d_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	lstm1d_forward_call_mode_th11(train, inst, time, start_seed);
};

void lstm1d_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	lstm1d_backward_call_mode_th11(train, inst, time, start_seed);
};