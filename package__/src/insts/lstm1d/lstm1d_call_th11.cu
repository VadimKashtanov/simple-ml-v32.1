#include "hip/hip_runtime.h"
#include "pkg_head/insts/lstm1d/lstm1d.cuh"

// =======================================================
// ===================== Use_t ===========================
// =======================================================

void lstm1d_use_call_mode_th11(Use_t * use, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint X=mdl->param[inst][0],		\
		 Y=mdl->param[inst][1],		\
		 istart=mdl->param[inst][2],\
		 ystart=mdl->param[inst][3],\
		 wstart=mdl->param[inst][4];

	//	Only th11 exists
	lstm1d_forward_th1x1<<<dim3(KERN_DIV(Y,16)),dim3(16)>>>(
		X, Y,
		time,
		mdl->total,
		istart, ystart, wstart,
		use->var, use->weight);
};

// =======================================================
// ==================== Forward ==========================
// =======================================================

void lstm1d_forward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint X=mdl->param[inst][0],			\
		 Y=mdl->param[inst][1],			\
		 istart=mdl->param[inst][2],	\
		 ystart=mdl->param[inst][3],	\
		 wstart=mdl->param[inst][4],	\
		 locdstart=mdl->param[inst][5],	\
		 drate=mdl->param[inst][6];

	uint total = mdl->total;
	uint wsize = mdl->weights;
	uint locdsize = mdl->locds;

	//	Only th11 exists
	lstm1d_forward_th1x1<<<dim3(KERN_DIV(Y,16),sets),dim3(16,1)>>>(
		X, Y,
		time,
		istart, ystart, wstart, locdstart,
		total, wsize, locdsize,
		train->_var, train->_weight, train->_locd,
		inst*start_seed, drate,
		train->sets);
};

// =======================================================
// ==================== Backward =========================
// =======================================================

void lstm1d_backward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	/*
		We could use __constant__[] and extern__shared__[] to call only one time from vram each locd
	*/
	Mdl_t * mdl = train->mdl;

	uint X=mdl->param[inst][0],			\
		 Y=mdl->param[inst][1],			\
		 istart=mdl->param[inst][2],	\
		 ystart=mdl->param[inst][3],	\
		 wstart=mdl->param[inst][4],	\
		 locdstart=mdl->param[inst][5],	\
		 drate=mdl->param[inst][6];

	uint total = mdl->total;
	uint wsize = mdl->weights;
	uint locdsize = mdl->locds;

	//	Only th11 exists
	lstm1d_backward_INPUT_th1x1<<<dim3(KERN_DIV(X,16),sets),dim3(16,1)>>>(		//backward Input (size = X)
		X, Y,
		time,
		istart, ystart, wstart, locdstart,
		total, wsize, locdsize,
		train->_var, train->_weight, train->_locd, train->_grad, train->_meand,
		inst*start_seed, drate,
		train->sets);

	if (time == 0) {
		lstm1d_backward_BIAS_ONLY_th1x1<<<dim3(KERN_DIV(Y,16),sets),dim3(16,1)>>>(	//backward output (size = Y)
			X, Y,
			time,
			istart, ystart, wstart, locdstart,
			total, wsize, locdsize,
			train->_var, train->_weight, train->_locd, train->_grad, train->_meand,
			inst*start_seed, drate,
			train->sets);
	} else {
		lstm1d_backward_H1_BIAS_th1x1<<<dim3(KERN_DIV(Y,16),sets),dim3(16,1)>>>(	//backward output (size = Y)
			X, Y,
			time,
			istart, ystart, wstart, locdstart,
			total, wsize, locdsize,
			train->_var, train->_weight, train->_locd, train->_grad, train->_meand,
			inst*start_seed, drate,
			train->sets);
	}
};