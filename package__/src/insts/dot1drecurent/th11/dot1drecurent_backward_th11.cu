#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot1drecurent/dot1drecurent_th11.cuh"

__global__
void dot1drecurent_backward_th1x1(
	uint Ax, uint At, uint Yx,
	uint activ,
	uint time,
	uint istart, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;
	float dlds;
	uint Apos, Wpos;

	if (y < Yx) {
		dlds = grad[time*sets*total + set*total + ystart + y] * locd[time*locdsize*sets + set*locdsize + locdstart + y];

		Apos = (time-At)*total*sets + set*total + istart;
		Wpos = ws*set + wstart + y;

		for (uint i=0; i < Ax; i++) {
			if (pseudo_randomf(Apos*seed) >= drop_rate) {
				atomicAdd(grad + Apos, weight[Wpos] * dlds);
				atomicAdd(meand + Wpos, var[Apos] * dlds);
			}

			Apos++;
			Wpos += Yx;
		}

		meand[Wpos + Yx] += dlds;
	}
}