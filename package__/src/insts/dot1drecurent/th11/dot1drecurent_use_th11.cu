#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot1drecurent/dot1drecurent_th11.cuh"

__global__
void dot1drecurent_use_th1x1(
	uint Ax, uint At, uint Yx,
	uint activ,
	uint time,
	uint total,
	uint input_start, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	float _tmp;

	uint Apos, Wpos;

	if (y < Yx) {

		_tmp = 0;

		Apos = (time-At)*total + istart;
		Wpos = wstart + y;

		for (uint i=0; i < Ax; i++) {
			_tmp += var[Apos] * weight[Wpos];
			Apos++;
			Wpos += Yx;
		}

		_tmp += weight[Wpos + Yx];	//==wstart + Ax*Yx + y      car on a deja +y, et on a += Ax*Yx (for i<Ax) {+=Yx}

		if (activ == 0)	_tmp = 1 / (1 + exp(-_tmp));
		else if (activ == 1) _tmp = tanh(_tmp);
		else if (activ == 2) _tmp = exp(-_tmp*_tmp);
		else _tmp *= (tmp > 0);

		var[time*total + ystart + y] = _tmp;
	}
};