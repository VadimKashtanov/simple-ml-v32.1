#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot1drecurent/dot1drecurent_th11.cuh"

__global__
void dot1drecurent_forward_th1x1(
	uint Ax, uint At, uint Yx,
	uint activ,
	uint time,
	uint input_start, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd,
	uint seed, float drop_rate,
	uint sets)
{
	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;
	float _tmp, _locd;

	uint Apos, Wpos;

	if (y < Yx) {
		_tmp = 0;

		Apos = (time-At)*total*sets + set*total + istart;
		Wpos = ws*set + wstart + y;

		for (uint i=0; i < Ax; i++) {
			if (pseudo_randomf(Apos*seed) >= drop_rate)
				_tmp += var[Apos] * weight[Wpos];
			Apos++;
			Wpos += Yx;
		}

		_tmp += weight[Wpos + Yx];	//==wstart + Ax*Yx + y      car on a deja +y, et on a += Ax*Yx (for i<Ax) {+=Yx}

		if (activ == 0)	{
			_tmp = 1 / (1 + exp(-_tmp));
			_locd = _tmp * (1 - _tmp);

		} else if (activ == 1) {
			_tmp = tanh(_tmp);
			_locd = 1 - _tmp*_tmp;

		} else if (activ == 2) {
			_locd = -2*_tmp;
			_tmp = exp(-_tmp*_tmp);
			_locd = _tmp * _locd;
		} else {
			_locd = (tmp > 0);
			_tmp = _tmp * _locd;
		}

		var[time*sets*total + set*total + ystart + y] = _tmp;
		locd[time*locdsize*sets + set*locdsize + locdstart + y] = _locd;
	}
}