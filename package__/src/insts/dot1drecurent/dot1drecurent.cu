#include "hip/hip_runtime.h"
#include "pkg_head/insts/dot1drecurent/dot1drecurent.cuh"

//			  0  1   2    3      4  5   6    7     8
//	Params : [Ax,At, Yx, activ, ist,yst,wst,lst, drate]
//	At - de combien de lignes on va en arriere. Si At=1 =>  A=A[t-1]

void dot1drecurent_check(uint * param) {
	if (param[0] == 0) 			 raise(SIGINT);
	if (param[2] == 0) 			 raise(SIGINT);
	if (param[3] >= ACTIV_FUNCS) raise(SIGINT);
	if (param[8] >100) 			 raise(SIGINT);
};

void dot1drecurent_cpu(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint Ax=mdl->param[inst][0],		\
		 At=mdl->param[inst][1],		\
		 Yx=mdl->param[inst][2],		\
		 acitv=mdl->param[inst][3]
		 istart=mdl->param[inst][4],	\
		 ystart=mdl->param[inst][5],	\
		 wstart=mdl->param[inst][6],	\
		 lstart=mdl->param[inst][7],	\
		 drate=mdl->param[inst][8];

	uint total = mdl->total;

	float * var = cpu->var;
	float * weight = mdl->weight;

	float tmp;

	uint Apos, Wpos;

	if (time - At >= 0) {
		for (uint y=0; y < Yx; y++) {
			_tmp = 0;

			Apos = (time-At)*total + istart;
			Wpos = wstart + y;

			for (uint i=0; i < Ax; i++) {
				_tmp += var[Apos] * weight[Wpos];
				Apos++;
				Wpos += Yx;
			}

			_tmp += weight[Wpos + Yx];	//==wstart + Ax*Yx + y      car on a deja +y, et on a += Ax*Yx (for i<Ax) {+=Yx}

			if (activ == 0)	_tmp = 1 / (1 + exp(-_tmp));
			else if (activ == 1) _tmp = tanh(_tmp);
			else if (activ == 2) _tmp = exp(-_tmp*_tmp);
			else _tmp *= (tmp > 0);

			var[time*total + ystart + y] = _tmp;
		}
	}
};

void dot1drecurent_use(Use_t * use, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint Ax=mdl->param[inst][0],		\
		 At=mdl->param[inst][1],		\
		 Yx=mdl->param[inst][2],		\
		 acitv=mdl->param[inst][3]
		 istart=mdl->param[inst][4],	\
		 ystart=mdl->param[inst][5],	\
		 wstart=mdl->param[inst][6],	\
		 lstart=mdl->param[inst][7],	\
		 drate=mdl->param[inst][8];

	uint total = mdl->total;
	uint wsize = mdl->weights;
	uint locdsize = mdl->locds;

	dot1drecurent_use_call_mode_th11(use, inst, time);
};

void dot1drecurent_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint Ax=mdl->param[inst][0],		\
		 At=mdl->param[inst][1],		\
		 Yx=mdl->param[inst][2],		\
		 acitv=mdl->param[inst][3]
		 istart=mdl->param[inst][4],	\
		 ystart=mdl->param[inst][5],	\
		 wstart=mdl->param[inst][6],	\
		 lstart=mdl->param[inst][7],	\
		 drate=mdl->param[inst][8];

	uint total = mdl->total;
	uint wsize = mdl->weights;
	uint locdsize = mdl->locds;

	dot1drecurent_forward_call_mode_th11(train, inst, time, start_seed);
};

void dot1drecurent_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint Ax=mdl->param[inst][0],		\
		 At=mdl->param[inst][1],		\
		 Yx=mdl->param[inst][2],		\
		 acitv=mdl->param[inst][3]
		 istart=mdl->param[inst][4],	\
		 ystart=mdl->param[inst][5],	\
		 wstart=mdl->param[inst][6],	\
		 lstart=mdl->param[inst][7],	\
		 drate=mdl->param[inst][8];

	uint total = mdl->total;
	uint wsize = mdl->weights;
	uint locdsize = mdl->locds;

	dot1drecurent_backward_call_mode_th11(train, inst, time, start_seed);
};