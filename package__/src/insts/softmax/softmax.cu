#include "hip/hip_runtime.h"
#include "pkg_head/insts/softmax.cuh"

void softmax_check(uint * param) {
	if (param[0] == 0) raise(SIGINT);
};

void softmax_cpu_call(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint 		len = mdl->param[inst][0],	\
		input_start	= mdl->param[inst][1],	\
			 ystart = mdl->param[inst][2];

	float __sum = 0;
	float tmp;

	uint vstart = time*mdl->total;
	float * var = cpu->var;

	for (uint i=0; i < len; i++) {
		tmp = exp(-var[vstart + input_start + i]);
		var[vstart + ystart + i] = exp(-var[vstart + input_start + i]);
		__sum += tmp;
	}

	for (uint i=0; i < len; i++)
		var[vstart + ystart + i] /= __sum;
};

void softmax_use_call(Use_t * use, uint inst, uint time) {
	softmax_use_call_mode_th32(use, inst, time);
};

void softmax_forward_call(Train_t * train, uint inst, uint time, uint start_seed) {
	softmax_forward_call_mode_th32(train, inst, time, start_seed);
};

void softmax_backward_call(Train_t * train, uint inst, uint time, uint start_seed) {
	softmax_backward_call_mode_th32(train, inst, time, start_seed);
};