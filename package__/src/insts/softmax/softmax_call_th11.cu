#include "pkg_head/insts/softmax/softmax_th32.cuh"

void softmax_use_call_mode_th32(Use_t * use, uint inst, uint time) {
	Mdl_t * mdl = use->mdl;

	uint len=mdl->param[inst][0],			\
		 input_start=mdl->param[inst][1],	\
		 ystart=mdl->param[inst][2];

	if (len <= 32) {
		softmax_use_th32<<<dim3(1),dim3(32)>>>(
			len,
			time,
			mdl->total,
			input_start, ystart,
			use->var);
		//hipDeviceSynchronize();
	} else {
		ERR("Can't handl more than 32 pixels for softmax")
	}
};

//======================== Train_t =======================

//-------------------------- forward ---------------------

void softmax_forward_call_mode_th32(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint len=mdl->param[inst][0],			\
		 input_start=mdl->param[inst][1],	\
		 ystart=mdl->param[inst][2],
		 locdstart=mdl->param[inst][3];

	if (len <= 32) {
		softmax_forward_th32<<<dim3(train->sets),dim3(32)>>>(
			len,
			time,
			mdl->total, mdl->locds,
			input_start, ystart, locdstart,
			train->sets,
			train->_var);
		//hipDeviceSynchronize();
	}
};

//-------------------------- backward ---------------------

void softmax_backward_call_mode_th32(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint len=mdl->param[inst][0],			\
		 input_start=mdl->param[inst][1],	\
		 ystart=mdl->param[inst][2],
		 locdstart=mdl->param[inst][3];

	if (len <= 32) {
		softmax_backward_th32<<<dim3(train->sets),dim3(32)>>>(
			len, 
			time,
			mdl->total, mdl->locds,
			input_start, ystart, locdstart,
			train->sets,
			train->_var, train->_grad);
		//hipDeviceSynchronize();
	}
};