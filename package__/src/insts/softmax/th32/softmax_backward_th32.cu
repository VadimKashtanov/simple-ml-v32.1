#include "hip/hip_runtime.h"
#include "pkg_head/insts/softmax.cuh"

__global__
void softmax_backward_th32(
	uint len,
	uint time,
	uint total, uint lsize,
	uint istart, uint ystart, uint lstart,
	uint sets,
	float * var, float * grad)
{
	uint pos = threadIdx.x;
	uint set = blockIdx.x;

	if (pos < len) {
		__shared__ float grads[32];
		__shared__ float ys[32];
		__shared__ float this_x_grad[32];

		uint start = time*sets*total + set*total + ystart + pos;

		grads[pos] = grad[start];	//the error
		ys[pos] = var[start];
		this_x_grad[pos] = 0;

		for (uint i=0; i < len; i++)
			atomicAdd(&grads[i], grads[i]*ys[i]*ys[pos]);
		atomicAdd(&this_x_grad[pos], -grads[pos]*pow(ys[pos],2));	//to avoid (if/else in for loop). We just exclude case
		atomicAdd(&this_x_grad[pos], grad[pos]*ys[pos]*(1 - ys[pos]));
		//
		start -= (ystart + pos);
		atomicAdd(&grad[start + istart + pos], this_x_grad[pos]);
	}
}