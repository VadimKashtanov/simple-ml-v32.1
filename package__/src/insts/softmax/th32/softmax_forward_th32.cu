#include "hip/hip_runtime.h"
#include "pkg_head/insts/softmax.cuh"

__global__
void softmax_forward_th32(
	uint len,
	uint time,
	uint total, uint lsize,
	uint istart, uint ystart, uint lstart,
	uint sets,
	float * var)
{
	uint pos = threadIdx.x;
	uint set = blockIdx.x;

	if (pos < len) {
		float exped = exp(-var[time*sets*total + set*total + istart + pos]);
		__shared__ float sum;
		if (pos == 0) sum = 0;
		__syncthreads();
		sum += exped;
		__syncthreads();
		var[time*sets*total + set*total + ystart + pos] = exped / sum;
	}
};