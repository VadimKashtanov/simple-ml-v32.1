#include "hip/hip_runtime.h"
#include "pkg_head/insts/softmax.cuh"

__global__
void softmax_use_th32(
	uint len,
	uint time,
	uint total,
	uint istart, uint ystart,
	float * var)
{
	uint pos = threadIdx.x;

	if (pos < len) {
		float exped = exp(-var[time*total + istart + pos]);
		__shared__ float sum;
		if (pos == 0) sum = 0;
		__syncthreads();
		atomicAdd(&sum, exped);
		__syncthreads();
		var[time*total + ystart + pos] = exped / sum;
	}
}