#include "hip/hip_runtime.h"
#include "pkg_head/insts/lstm2d/lstm2d.cuh"

//			   0  1  2     3      4      5       6        7 
//Arguments = [Ax,Ay,Bx, istart,ystart,wstart,locdstart, drate]

void lstm2d_check(uint * param) {
	if (param[0] == 0) raise(SIGINT);
	if (param[1] == 0) raise(SIGINT);
	if (param[2] == 0) raise(SIGINT);
	if (param[7] >100) raise(SIGINT);
};

void lstm2d_cpu(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint Ax=mdl->param[inst][0],		\
		 Ay=mdl->param[inst][1],		\
		 Bx=mdl->param[inst][2],		\
		 istart=mdl->param[inst][3],	\
		 ystart=mdl->param[inst][4],	\
		 wstart=mdl->param[inst][5],	\
		 locdstart=mdl->param[inst][6],	\
		 drate=mdl->param[inst][7];

	uint total = mdl->total;

	uint inp = total*time + istart;
	uint W = wstart;
	uint out = total*time + ystart;

	uint _W = Bx * Ax;
	uint _U = Bx * Bx;
	uint _B = Bx * Ay;

	uint lineW = _W + _U + _B;

	float * var = cpu->var;
	float * weight = mdl->weight;

	float f0,f1,f2,g0;
	float xval;
	float e,e_1,h;

	for (uint x=0; x < Bx; x++) {
		for (uint y=0; y < Ay; y++) {
			//	Compute f0,f1,f2
			f0 = 0; f1 = 0; f2 = 0; g0 = 0;

			//x@.W
			for (uint k=0; k < Ax; k++) {
				xval = var[total*time + istart + (y*Ax + k)];
				f0 += weight[wstart + (k*Bx + x)] * xval;
				f1 += weight[wstart + lineW + (k*Bx + x)] * xval;
				f2 += weight[wstart + 2*lineW + (k*Bx + x)] * xval;
				g0 += weight[wstart + 3*lineW + (k*Bx + x)] * xval;
			}

			//h[-1]@.U
			if (time > 0) {
				for (uint k=0; k < Bx; k++) {
					xval = var[total*(time-1) + ystart + (y*Bx + k)];
					f0 += weight[wstart + _W + (k*Bx + x)] * xval;
					f1 += weight[wstart + lineW + _W + (k*Bx + x)] * xval;
					f2 += weight[wstart + 2*lineW + _W + (k*Bx + x)] * xval;
					g0 += weight[wstart + 3*lineW + _W + (k*Bx + x)] * xval;
				}
			}

			f0 = logistic(f0 + weight[wstart + _W + _U + (y*Bx + x)]);
			f1 = logistic(f1 + weight[wstart + lineW + _W + _U + (y*Bx + x)]);
			f2 = logistic(f2 + weight[wstart + 2*lineW + _W + _U + (y*Bx + x)]);
			g0 = tanh(g0 + weight[wstart + 3*lineW +_W + _U + (y*Bx + x)]);

			if (time > 0) e_1 = var[total*(time-1) + ystart + (y*Bx + x)];
			else e_1 = 0;

			e = f0 * e_1 + f1 * g0;
			h = f2 * e;

			var[total*time + ystart + (y*Bx + x)] = e;
			var[total*time + ystart + Bx*Ay + (y*Bx + x)] = h;
		}
	}
};

void lstm2d_use(Use_t * use, uint inst, uint time) {
	lstm2d_use_call_mode_th11(use, inst, time);
};

void lstm2d_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	lstm2d_forward_call_mode_th11(train, inst, time, start_seed);
};

void lstm2d_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	lstm2d_backward_call_mode_th11(train, inst, time, start_seed);
};