#include "hip/hip_runtime.h"
#include "pkg_head/insts/lstm2d.cuh"

__global__
void lstm2d_use_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint time,
	uint total,
	uint istart, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < Bx && y < Ay)
	{
		uint inp = total*time + istart;
		uint W = wstart;
		uint out = total*time + ystart;
		uint locdpos = locds*time + locdstart;

		uint _W = Bx * Ax;
		uint _U = Bx * Bx;
		uint _B = Bx * Ay;

		uint lineW = _W + _U + _B;

		uint vpos, wpos;

		// f0,f1,f2 = logistic(x@W + h[-1]@U + B)
		// g0 	  = tanh 	(x@W + h[-1]@U + B)
		float f0=0,f1=0,f2=0,g0=0;

		float tmpt;

		// .W
		for (uint k=0; k < Ax; k++) {	//for all in INPUT
			//	Positions
			vpos = inp + (y*Ax + k);

			//
			wpos = k*Bx + x;

			tmpt = var[vpos];
			f0 += tmpt * weight[W + 0*lineW + wpos];
			f1 += tmpt * weight[W + 1*lineW + wpos];
			f2 += tmpt * weight[W + 2*lineW + wpos];
			g0 += tmpt * weight[W + 3*lineW + wpos];
		}

		// .U
		if (time > 0) {
			for (uint k=0; k < Bx; k++) {
				vpos = total*(time-1) + ystart + (Bx*Ay) + y*Bx + k;	///h[-1]
				wpos = _W + k*Bx + x;

				tmpt = var[vpos];
				f0 += tmpt * weight[W + 0*lineW + wpos];
				f1 += tmpt * weight[W + 1*lineW + wpos];
				f2 += tmpt * weight[W + 2*lineW + wpos];
				g0 += tmpt * weight[W + 3*lineW + wpos];
			}
		}

		// .B
		wpos = _W + _U + y*Bx + x;
		f0 += w[W + 0*lineW + wpos];
		f1 += w[W + 1*lineW + wpos];
		f2 += w[W + 2*lineW + wpos];
		g0 += w[W + 3*lineW + wpos];

		// activ(_sum)
		f0 = logistic(f0);
		f1 = logistic(f1);
		f2 = logistic(f2);
		g0 = tanh(g0);

		// e = f0 * e[-1] + f1 * g0
		// l - 1 have to be >= 0
		float e_1;
		if (time > 0) e_1 = var[total*(time-1) + ystart + y*Bx + x];
		else e_t = 0;
		
		float e = f0*e_1 + f1*g0;
		float h = f2 * e;

		locd[locdpos + 0*Bx*Ay + (y*Bx + x)] = f2*e_1*( f0*(1 - f0) );		//	f0
		locd[locdpos + 1*Bx*Ay + (y*Bx + x)] = f2*g0*( f1*(1 - f1) );	//	f1
		locd[locdpos + 2*Bx*Ay + (y*Bx + x)] = e*( f2*(1 - f2) );	//	f2
		lcod[locdpos + 3*Bx*Ay + (y*Bx + x)] = f2*f1*( 1 - g0*g0);	//	g0

		var[out + 0*Bx*Ay + y*Bx + x] = e;
		var[out + 1*Bx*Ay + y*Bx + x] = h;
	};
};