#include "hip/hip_runtime.h"
#include "pkg_head/insts/lstm2d.cuh"

__global__
void lstm2d_forward_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd,
	uint seed, float drop_rate,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;
	
	if (x < Bx && y < Ay)
	{
		uint inp = total*sets*time + total*set + istart;
		uint W = ws*set + wstart;
		uint out = total*sets*time + total*set + ystart;
		uint locdpos = lsize*sets*time + lsize*set + locdstart;

		uint _W = Bx * Ax;
		uint _U = Bx * Bx;
		uint _B = Bx * Ay;

		uint lineW = _W + _U + _B;

		uint vpos, wpos;

		// f0,f1,f2 = logistic(x@W + h[-1]@U + B)
		// g0 	  = tanh 	(x@W + h[-1]@U + B)
		float f0=0,f1=0,f2=0,g0=0;

		float tmpt;

		// .W
		for (uint k=0; k < Ax; k++) {	//for all in INPUT
			//	Positions
			vpos = inp + (y*Ax + k);

			//	Drop
			if (pseudo_randomf(seed + vpos) > drop_rate) {
				
				//
				wpos = k*Bx + x;

				tmpt = var[vpos];
				f0 += tmpt * weight[W + 0*lineW + wpos];
				f1 += tmpt * weight[W + 1*lineW + wpos];
				f2 += tmpt * weight[W + 2*lineW + wpos];
				g0 += tmpt * weight[W + 3*lineW + wpos];
			}
		}

		// .U
		if (time > 0) {
			for (uint k=0; k < Bx; k++) {
				vpos = total*sets*(time-1) + total*set + ystart + (Bx*Ay) + y*Bx + k;	///h[-1]
				wpos = _W + k*Bx + x;

				tmpt = var[vpos];
				f0 += tmpt * weight[W + 0*lineW + wpos];
				f1 += tmpt * weight[W + 1*lineW + wpos];
				f2 += tmpt * weight[W + 2*lineW + wpos];
				g0 += tmpt * weight[W + 3*lineW + wpos];
			}
		}

		// .B
		wpos = _W + _U + y*Bx + x;
		f0 += w[W + 0*lineW + wpos];
		f1 += w[W + 1*lineW + wpos];
		f2 += w[W + 2*lineW + wpos];
		g0 += w[W + 3*lineW + wpos];

		// activ(_sum)
		f0 = logistic(f0);
		f1 = logistic(f1);
		f2 = logistic(f2);
		g0 = tanh(g0);

		// e = f0 * e[-1] + f1 * g0
		// l - 1 have to be >= 0
		float e_1;
		if (time > 0) e_1 = var[total*sets*(time-1) + total*set + ystart + (y*Bx + x)];
		else e_t = 0;
		
		float e = f0*e_1 + f1*g0;
		float h = f2 * e;

		//	n*Bx*Ay car on stoque 4x la matrice Bx*Ay des derives locales pour la derivee en chaine de f0, f1, f2 et g0
		locd[locdpos + 0*Bx*Ay + (y*Bx + x)] = f0;//f2*e_1*( f0*(1 - f0) );		//	f0
		locd[locdpos + 1*Bx*Ay + (y*Bx + x)] = f1;//f2*g0*( f1*(1 - f1) );	//	f1
		locd[locdpos + 2*Bx*Ay + (y*Bx + x)] = f2;//e*( f2*(1 - f2) );	//	f2
		locd[locdpos + 3*Bx*Ay + (y*Bx + x)] = g0;//f2*f1*( 1 - g0*g0);	//	g0

		//	On stoque dans le output 2 matrices Bx*Ay  ou il y a `e` et `h`.
		//	`h` est le resultat du LSTM
		//	`e` est juste utilise pour avoire la ligne d'apres le e[-1]
		var[out + 0*Bx*Ay + (y*Bx + x)] = e;
		var[out + 1*Bx*Ay + (y*Bx + x)] = h;
	};
};