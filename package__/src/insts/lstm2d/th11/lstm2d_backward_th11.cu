#include "hip/hip_runtime.h"
#include "pkg_head/insts/lstm2d.cuh"

/*			  =======
			  |     |
			  |		|
			  |	.W	|
			  |		|
			  |		|
			  =======
============= =======
|	.input	| | 	|	input@W
============= =======
				 +
			  =======
			  |	.U  |
			  |	    |
			  =======
	  ======= =======
	  |h[-1]| |		|  h[-1]@U
	  ======= =======
				 +
			  =======
			  |	.B	|
			  =======
*/

/*	We could use atomicAdd with 1 direct backward function

*/

__global__  //ca veut dire que le kernel est position sur les cooredonne de l'input, et chaque kernel est associe a un pixel de l'input. Puis on backward on ligne verticale de .W
void lstm2d_backward_INPUT_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	/*	Backward grad(input)
		meand(.W) of f0,f1,f2,g0
	*/

	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	uint ipos = total*sets*time + total*set + istart + (y*Ax + x);

	//input = Ax*Ay, and the (x,y) pixel is in input. Then we backward .W and this pixel gradient
	if (x < Ax && y < Ay && pseudo_randomf(seed + ipos) > drop_rate) {	//if input[x] is droped, following will be *0

		float grad_input_compute = 0;	//_INPUT_ et _H1_ ajoutent un gradient a input[x]

		uint W = ws*set + wstart;
		uint out = total*sets*time + total*set + ystart;
		uint locdpos = lsize*sets*time + lsize*set + lstart;

		uint _W = Bx * Ax;
		uint _U = Bx * Bx;
		uint _B = Bx * Ay;

		uint lineW = _W + _U + _B;

		//uint vpos = total*sets*time + set*total + istart + x;
		float xval = var[ipos];

		float chain_deriv;
		float dH,f0,f1,f2,g0,de;
		float dsf0, dsf1, dsf2, dsg0;

		uint wpos, epos, e_1pos, hpos, outpos;	//wpos   = position du weight en question
												//epos,e_1pos,hpos = output `e` ou `h` (car output = `e` + `h`). e_1 est e[-1]
												//outpos = (y*Bx+k) juste pour calculer de quel pixel de Y nous prenon le locd (car on backward chaque colone de output mais les weights d'une meme ligne) 

		//	Backward W
		for (uint k=0; k < Bx; k++) {	//[ w0 w1 w2 w3 ... wn]	une ligne du .W (la premiere par exemple)
										//car inp[x] est multiplice par `w[x*Bx + k] for k in Bx`  ou Bx==Y

			outpos = y*Bx + k;

			epos = out + outpos;
			e_1pos = total*sets*(time-1) + total*set + ystart + outpos; //if l == 0 , e_1pos <= 0
			hpos = out + Bx*Ay + outpos;

			dH = grad[hpos];

			f0 = locd[locdpos + 0*Bx*Ay + outpos];// * dH;
			f1 = locd[locdpos + 1*Bx*Ay + outpos];// * dH;
			f2 = locd[locdpos + 2*Bx*Ay + outpos];// * dH;
			g0 = lcod[locdpos + 3*Bx*Ay + outpos];// * dH;

			de = grad[epos] + dH * f2;	//grad(e) += dH*f2

			grad[epos] = de;

			//if time > 0:
			grad[e_1pos] += de * f0;		//we can't store only 4 locds, because how will we get de*f0 ?

			dsf0 = de * var[e_1pos] * f0 * (1 - f0);
			dsf1 = de * g0 * f1 * (1 - f1);
			dsf2 = dH * e * f2 * (1 - f2);
			dsg0 = de * f1 * (1 - g0*g0);

			//	f0
			wpos = ws*set + wstart + 0*lineW + (x*Y + k);			//on met a jour que .W pas .U no .B
			//meand[wpos] += dsf0 * xvalue;
			atomicAdd(meand + wpos, dsf0 * xvalue);
			grad_input_compute += dsf0 * weight[wpos];

			//	f1
			wpos = ws*set + wstart + 1*lineW + (x*Y + k);			//on met a jour que .W pas .U no .B
			//meand[wpos] += dsf1 * xvalue;
			atomicAdd(meand + wpos, dsf1 * xvalue);
			grad_input_compute += dsf1 * weight[wpos];

			//	f2
			wpos = ws*set + wstart + 2*lineW + (x*Y + k);			//on met a jour que .W pas .U no .B
			//meand[wpos] += dsf2 * xvalue;
			atomicAdd(meand + wpos, dsf2 * xvalue);
			grad_input_compute += dsf2 * weight[wpos];

			//	g0
			wpos = ws*set + wstart + 3*lineW + (x*Y + k);			//on met a jour que .W pas .U no .B
			//meand[wpos] += dsg0 * xvalue;
			atomicAdd(meand + wpos, dsg0 * xvalue);
			grad_input_compute += dsg0 * weight[wpos];
		}

		//	Backward input
		grad[ipos] += grad_input_compute;
		//atomicAdd(grad + vpos, grad_input_compute);
	}
}

__global__
void lstm2d_backward_H1_BIAS_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint time,
	uint input_start, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	//
	//	h[-1] @ .U
	//

	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint x = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (x < Bx && y < Ay) {	//Only input is under drop. h is an output. It's values, could be droped, but in an other instruction

		float grad_H1_compute = 0;	//_INPUT_ et _H1_ ajoutent un gradient a input[x]

		uint h1pos = total*sets*(time-1) + set*total + istart + (y*Bx + x);	//h[-1] pos
		float h1val = var[h1pos];

		uint W = ws*set + wstart;
		uint out = total*sets*time + total*set + ystart;
		uint locdpos = lsize*sets*time + lsize*set + lstart;

		uint _W = Bx * Ax;
		uint _U = Bx * Bx;
		uint _B = Bx * Ay;

		uint lineW = _W + _U + _B;

		float chain_deriv;
		float _grad;	//of h[t]

		uint wpos;

		for (uint k=0; k < Bx; k++) {	//[ w0 w1 w2 w3 ... wn]	une ligne du .W (la premiere par exemple)
										//car inp[x] est multiplice par `w[x*Bx + k] for k in Bx`  ou Bx==Y
										//en fait k est la colone de la matrice. la ligne est `y` du kernel
										//et le `x` du kernel determine le pixel `h[-1]` et la ligne dans .U 

			outpos = y*Bx + k;
			
			epos = out + outpos;
			e_1pos = total*sets*(time-1) + total*set + ystart + outpos; //if l == 0 , e_1pos <= 0
			hpos = out + Bx*Ay + outpos;

			dH = grad[hpos];

			f0 = locd[locdpos + 0*Bx*Ay + outpos];// * dH;
			f1 = locd[locdpos + 1*Bx*Ay + outpos];// * dH;
			f2 = locd[locdpos + 2*Bx*Ay + outpos];// * dH;
			g0 = lcod[locdpos + 3*Bx*Ay + outpos];// * dH;

			de = grad[epos] + dH * f2;	//grad(e) += dH*f2

			grad[epos] = de;

			//if time > 0:
			grad[e_1pos] += de * f0;

			dsf0 = de * var[e_1pos] * f0 * (1 - f0);
			dsf1 = de * g0 * f1 * (1 - f1);
			dsf2 = dH * e * f2 * (1 - f2);
			dsg0 = de * f1 * (1 - g0*g0);

			//	f0
			wpos = W + 0*lineW + _W + (x*Bx + k);					//on met a jour que .U pas .W no .B
			//meand[wpos] += dsf0 * h1val;
			atomicAdd(meand + wpos, dsf0 * h1val);
			grad_h1_compute += dsf0 * weight[wpos];

			//	f1
			wpos = W + 1*lineW + _W + (x*Bx + k);			//on met a jour que .U pas .W no .B
			//meand[wpos] += dsf1 * h1val;
			atomicAdd(meand + wpos, dsf1 * h1val);
			grad_h1_compute += dsf1 * weight[wpos];

			//	f2
			wpos = W + 2*lineW + _W + (x*Bx + k);			//on met a jour que .U pas .W no .B
			//meand[wpos] += dsf2 * h1val;
			atomicAdd(meand + wpos, dsf2 * h1val);
			grad_h1_compute += dsf2 * weight[wpos];
		
			//	g0
			wpos = W + 3*lineW + _W + (x*Bx + k);			//on met a jour que .U pas .W no .B
			//meand[wpos] += dsg0 * h1val;
			atomicAdd(meand + wpos, dsg0 * h1val);
			grad_h1_compute += dsg0 * weight[wpos];
		}

		//	Backward h[-1]
		grad[h1pos] += grad_input_compute;
		//atomicAdd(grad + vpos, grad_input_compute);

		//  ============================================
		//	Backward .B
		//	Vu que la grille est de <<<Bx,Ay>>> on en profite car .B l'est aussi
		//	Au lieu de cree un autre fonction qui compute le gradient de .B, on le fait directe ici.	
		//

		outpos = y*Bx + x;

		epos = out + outpos;
		e_1pos = total*sets*(time-1) + total*set + ystart + outpos; //if l == 0 , e_1pos <= 0
		hpos = out + Bx*Ay + outpos;

		dH = grad[hpos];

		f0 = locd[locdpos + 0*Bx*Ay + outpos];// * dH;
		f1 = locd[locdpos + 1*Bx*Ay + outpos];// * dH;
		f2 = locd[locdpos + 2*Bx*Ay + outpos];// * dH;
		g0 = lcod[locdpos + 3*Bx*Ay + outpos];// * dH;

		de = grad[epos] + dH * f2;	//grad(e) += dH*f2
		grad[epos] = de;

		//if time > 0:
		grad[e_1pos] += de * f0;

		dsf0 = de * var[e_1pos] * f0 * (1 - f0);
		dsf1 = de * g0 * f1 * (1 - f1);
		dsf2 = dH * e * f2 * (1 - f2);
		dsg0 = de * f1 * (1 - g0*g0);

		//	f0
		meand[W + 0*lineW + _W + _U + (x*Y + k)] += dsf0;

		//	f1
		meand[W + 1*lineW + _W + _U + (x*Y + k)] += dsf1;

		//	f2
		meand[W + 2*lineW + _W + _U + (x*Y + k)] += dsf2;

		//	g0
		meand[W + 3*lineW + _W + _U + (x*Y + k)] += dsg0;
	}
};

__global__
void lstm2d_backward_BIAS_ONLY_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	//
	//	h[-1] @ .U
	//

	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint x = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (x < Bx && y < Ay) {	//Only input is under drop. h is an output. It's values, could be droped, but in an other instruction

		uint W = ws*set + wstart;
		uint out = total*sets*time + total*set + ystart;
		uint locdpos = lsize*sets*time + lsize*set + lstart;

		uint _W = Bx * Ax;
		uint _U = Bx * Bx;
		uint _B = Bx * Ay;

		uint lineW = _W + _U + _B;

		float chain_deriv;
		float _grad;	//of h[t]

		//  ============================================
		//	Backward .B
		//	Vu que la grille est de <<<Bx,Ay>>> on en profite car .B l'est aussi
		//	Au lieu de cree un autre fonction qui compute le gradient de .B, on le fait directe ici.	
		//
		
		outpos = y*Bx + x;

		epos = out + outpos;
		e_1pos = total*sets*(time-1) + total*_set + ystart + outpos; //if l == 0 , e_1pos <= 0
		hpos = out + Bx*Ay + outpos;

		dH = grad[hpos];

		f0 = locd[locdpos + 0*Bx*Ay + outpos];// * dH;
		f1 = locd[locdpos + 1*Bx*Ay + outpos];// * dH;
		f2 = locd[locdpos + 2*Bx*Ay + outpos];// * dH;
		g0 = lcod[locdpos + 3*Bx*Ay + outpos];// * dH;

		de = grad[epos] + dH * f2;	//grad(e) += dH*f2
		grad[epos] = de;

		//if time > 0:
		grad[e_1pos] += de * f0;

		dsf0 = de * var[e_1pos] * f0 * (1 - f0);
		dsf1 = de * g0 * f1 * (1 - f1);
		dsf2 = dH * e * f2 * (1 - f2);
		dsg0 = de * f1 * (1 - g0*g0);

		//	f0
		meand[W + 0*lineW + _W + _U + (x*Y + k)] += dsf0;

		//	f1
		meand[W + 1*lineW + _W + _U + (x*Y + k)] += dsf1;

		//	f2
		meand[W + 2*lineW + _W + _U + (x*Y + k)] += dsf2;

		//	g0
		meand[W + 3*lineW + _W + _U + (x*Y + k)] += dsg0;
	}
};
