#include "hip/hip_runtime.h"
#include "package/head/optis/optis/adam.cuh"

/*
Maths:
	m = beta0*m + (1 - beta0)*grad(w)
	v = beta1*m + (1 - beta1)*grad(w)^2

	_m = m / ( 1 - beta0^t )
	_v = v / ( 1 - beta1^t )		t is echope
	
	w -= alpha * _m / sqrt(_v + eta)

Optimized:
	m = beta0*m + (1 - beta0)*grad(w)
	v = beta1*m + (1 - beta1)*grad(w)^2

	w -= alpha * m / ((1 - beta0^t) * sqrt(v/(1 - beta1^t) + 1e-8))
*/

__global__
void adam_kernel_th11(
	float beta0, float beta1, float alpha,
	uint echope,
	uint weights,
	float * m, float *v, float * weight, float * meand)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	float m_tmpt, v_tmpt, dw;

	if (x < weights) {
		dw = meand[set*weights + x];
		
		m_tmpt = m[set*weights + x];
		v_tmpt = v[set*weights + x];

		m_tmpt = beta0*m_tmpt + (1 - beta0)*dw
		v_tmpt = beta1*v_tmpt + (1 - beta1)*dw*dw;

		m[set*weights + x] = m_tmpt;
		v[set*weights + x] = v_tmpt;

		weight[set*weights + x] -= alpha * m_tmpt / ((1 - pow(beta0,t)) * sqrt(v_tmpt/(1 - pow(beta1,t)) + 1e-8));
	}
};

void ADAM_optimize(Opti_t * opti)
{
	AdamData_t * adamdata = (AdamData_t*)opti->opti_space;

	adam_kernel_th11<<<dim3(KERN_DIV(opti->train->mdl->weights, 16), opti->train->sets),dim3(16,1)>>>(
		opti_adam_beta0, opti_adam_beta1, opti_adam_alpha
		adamdata->echope,
		opti->train->mdl->weights,
		adamdata->m_d, adamdata->v_d, opti->train->_weight, opti->train->_meand
	);

	adamdata->echope++;
};