#include "package/head/optis/optis/adam.cuh"

void * ADAM_space_mk(Opti_t * opti) {
	float * m_d * v_d;

	SAFE_CUDA(hipMalloc((void**)&m_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	SAFE_CUDA(hipMalloc((void**)&v_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	SAFE_CUDA(hipMemset(m_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	SAFE_CUDA(hipMemset(v_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	AdamData_t * ret = (AdamData_t*)malloc(sizeof(AdamData_t));

	ret->m_d = m_d;
	ret->v_d = v_d;

	ret->echope = 0;

	return (void*)ret;
};

void ADAM_free(Opti_t * opti) {
	SAFE_CUDA(hipFree((AdamData_t*)opti->opti_space->m_d))
	SAFE_CUDA(hipFree((AdamData_t*)opti->opti_space->v_d))
	free((AdamData_t*)opti->opti_space);
};