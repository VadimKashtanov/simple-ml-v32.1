#include "hip/hip_runtime.h"
#include "package/head/optis/optis/adamax.cuh"

/*
	m = beta0*m + (1 - beta0)*grad(w)
	u = max(beta1 * u, abs(grad(w)))
	
	w -= (alpha/(1 - beta1^t)) * m / u
*/

__global__
void adamax_kernel_th11(
	float alpha, float beta0, float beta1,
	uint weights,
	float * m, float *u, float * weight, float * meand)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	float m_tmpt, u_tmpt, dw;

	if (x < weights) {
		dw = meand[set*weights + x];
		
		m_tmpt = m[set*weights + x];
		u_tmpt = u[set*weights + x];

		m_tmpt = beta0*m_tmpt + (1 - beta0)*dw;
		u_tmpt = max(beta1 * u_tmpt, abs(dw));

		m[set*weights + x] = m_tmpt;
		u[set*weights + x] = u_tmpt;

		weight[set*weights + x] -= (alpha / (1 - pow(beta0, t))) * m_tmpt/u_tmpt;
	}
};

void ADAMAX_optimize(Opti_t * opti)
{
	AdamaxData_t * adamaxdata = (AdamaxData_t*)opti->opti_space;

	adamax_kernel_th11<<<dim3(KERN_DIV(opti->train->mdl->weights, 16), opti->train->sets),dim3(16,1)>>>(
		opti_adamax_alpha, opti_adamax_beta0, opti_adamax_beta1,
		adamaxdata->echopes,
		opti->train->mdl->weights,
		adamaxdata->m_d, adamaxdata->u_d, opti->train->_weight, opti->train->_meand
	);

	adamaxdata->echopes++;
};