#include "package/head/optis/optis/adamax.cuh"

void * ADAMAX_space_mk(Opti_t * opti) {
	float * m_d * u_d;

	SAFE_CUDA(hipMalloc((void**)&m_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	SAFE_CUDA(hipMalloc((void**)&u_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	SAFE_CUDA(hipMemset(m_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	SAFE_CUDA(hipMemset(u_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	AdamaxData_t * ret = (AdamaxData_t*)malloc(sizeof(AdamaxData_t));

	ret->m_d = m_d;
	ret->u_d = u_d;

	ret->echopes = 0;

	return (void*)ret;
};

void ADAMAX_free(Opti_t * opti) {
	SAFE_CUDA(hipFree((AdamaxData_t*)opti->opti_space->u_d))
	SAFE_CUDA(hipFree((AdamaxData_t*)opti->opti_space->m_d))
	free((AdamaxData_t*)opti->opti_space);
};