#include "hip/hip_runtime.h"
#include "package/head/optis/optis/rmsprop.cuh"

__global__
void RMSPROP_kernel_th11(
	float alpha, float beta,
	uint weights,
	float * v, float * weight, float * meand)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	float _v, dw;

	if (x < weights) {
		dw = meand[set*weights + x];
		_v = beta * v[set*weights + x] + (1-beta) * pow(dw,2);
		v[set*weights + x] = _v;
		weight[set*weights + x] -= alpha * dw * pow(_v + 1e-8, -0.5);	//eta = 1e-8
	}
};

void RMSPROP_optimize(Opti_t * opti)
{
	rmsprop_kernel_th11<<<dim3(KERN_DIV(opti->train->mdl->weights, 16), opti->train->sets),dim3(16,1)>>>(
		opti_momentum_alpha, opti_momentum_moment,
		opti->train->mdl->weights,
		(float*)opti->opti_space, opti->train->_weight, opti->train->_meand
	);
};