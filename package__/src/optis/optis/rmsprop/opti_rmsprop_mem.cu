#include "package/head/optis/optis/rmsprop.cuh"

void * RMSPROP_space_mk(Opti_t * opti) {
	float * v0_d;//, * v1_d;

	SAFE_CUDA(hipMalloc((void**)&v0_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	//SAFE_CUDA(hipMalloc((void**)&v1_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	SAFE_CUDA(hipMemset(v0_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	//SAFE_CUDA(hipMemset(v1_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	//RMSprop_data_t * ret = (RMSprop_data_t*)malloc(sizeof(RMSprop_data_t));

	//ret->v0_d = v0_d;
	//ret->v1_d = v1_d;

	//return (void*)ret;
	return (void*)v0_d;
};

void RMSPROP_free(Opti_t * opti) {
	//SAFE_CUDA(hipFree((RMSprop_data_t*)opti->opti_space->v0_d))
	//SAFE_CUDA(hipFree((RMSprop_data_t*)opti->opti_space->v1_d))
	//free((RMSprop_data_t*)opti->opti_space);
	SAFE_CUDA(hipFree((float*)opti->opti_space))
};