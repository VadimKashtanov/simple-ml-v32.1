#include "hip/hip_runtime.h"
#include "package/head/optis/optis/sgd/sgd.cuh"

__global__
void sgd_kernel_th11(
	float sgd_alpha,
	uint weights,
	float * weight, float * meand)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	if (x < weights)
		weight[set*weights + x] -= sgd_alpha * meand[set*weights + x];
};

void SGD_optimize(Opti_t * opti)
{
	sgd_kernel_th11<<<dim3(KERN_DIV(opti->train->mdl->weights, 16), opti->train->sets),dim3(16,1)>>>(
		opti_sgd_alpha,
		opti->train->mdl->weights,
		opti->train->_weight, opti->train->_meand
	);
};