#include "hip/hip_runtime.h"
#include "package/head/optis/optis/momentum.cuh"

__global__
void momentum_kernel_th11(
	float alpha, float moment,
	uint weights,
	float * v, float * weight, float * meand)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	float _v;

	if (x < weights) {
		_v = moment * v[set*weights + x] - alpha * meand[set*weights + x];
		v[set*weights + x] = _v;
		weight[set*weights + x] += _v;
	}
};

void MOMENTUM_optimize(Opti_t * opti)
{
	momentum_kernel_th11<<<dim3(KERN_DIV(opti->train->mdl->weights, 16), opti->train->sets),dim3(16,1)>>>(
		opti_momentum_alpha, opti_momentum_moment,
		opti->train->mdl->weights,
		(float*)opti->opti_space, opti->train->_weight, opti->train->_meand
	);
};