#include "package/head/optis/optis/momentum.cuh"

void * MOMENTUM_space_mk(Opti_t * opti) {
	float * ret_d;

	SAFE_CUDA(hipMalloc((void**)&ret_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	SAFE_CUDA(hipMemset(ret_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	return (void*)ret_d;
};

void MOMENTUM_free(Opti_t * opti) {
	SAFE_CUDA(hipFree((float*)opti->opti_space))
};