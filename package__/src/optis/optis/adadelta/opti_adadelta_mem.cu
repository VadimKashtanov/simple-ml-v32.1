#include "package/head/optis/optis/adadelta.cuh"

void * ADADELTA_space_mk(Opti_t * opti) {
	float * m_d * v_d;

	SAFE_CUDA(hipMalloc((void**)&m_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	SAFE_CUDA(hipMalloc((void**)&v_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	SAFE_CUDA(hipMemset(m_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	SAFE_CUDA(hipMemset(v_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	AdadeltaData_t * ret = (AdadeltaData_t*)malloc(sizeof(AdadeltaData_t));

	ret->m_d = m_d;
	ret->v_d = v_d;

	return (void*)ret;
};

void ADADELTA_free(Opti_t * opti) {
	SAFE_CUDA(hipFree((AdadeltaData_t*)opti->opti_space->v_d))
	SAFE_CUDA(hipFree((AdadeltaData_t*)opti->opti_space->m_d))
	free((AdadeltaData_t*)opti->opti_space);
};