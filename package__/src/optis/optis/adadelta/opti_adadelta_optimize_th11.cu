#include "hip/hip_runtime.h"
#include "package/head/optis/optis/adadelta.cuh"

/*
	m = beta0*m + (1 - beta0)*grad(w)^2
	delta_w = - sqrt(v + 1e-8) / sqrt(m + 1e-8)
	v = beta1*v + (1 - beta1)*delta_w^2

	w = delta_w
*/

__global__
void adadelta_kernel_th11(
	float beta0, float beta1,
	uint weights,
	float * m, float *v, float * weight, float * meand)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	float m_tmpt, old_v_tmpt, delta_w, dw;

	//	dw - derivative of w
	//	delta_w - change in w

	if (x < weights) {
		dw = meand[set*weights + x];
		
		m_tmpt = m[set*weights + x];
		old_v_tmpt = v[set*weights + x];

		m_tmpt = beta0*m_tmpt + (1 - beta0)*dw;
		m[set*weights + x] = m_tmpt;

		delta_w = - sqrt(old_v_tmpt + 1e-8) / sqrt(m_tmpt + 1e-8);

		v[set*weights + x] = beta1*old_v_tmpt + (1 - beta1)*delta_w*delta_w;

		weight[set*weights + x] += delta_w;	//le `-` est deja dans le delta_w
	}
};

void ADADELTA_optimize(Opti_t * opti)
{
	AdadeltaData_t * adadeltadata = (AdadeltaData_t*)opti->opti_space;

	adadelta_kernel_th11<<<dim3(KERN_DIV(opti->train->mdl->weights, 16), opti->train->sets),dim3(16,1)>>>(
		opti_adadelta_beta0, opti_adadelta_beta1,
		opti->train->mdl->weights,
		adadeltadata->m_d, adadeltadata->v_d, opti->train->_weight, opti->train->_meand
	);
};