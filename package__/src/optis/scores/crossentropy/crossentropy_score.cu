#include "hip/hip_runtime.h"
#include "package/head/optis/scores/crossentropy.cuh"

#ifndef ln
	#define ln(x) log(x)
#endif 	// ln = log(x, e) but log(x, e) == log(e)  in C
		//	It's more for the form and clarity than for optimisation 

#define CROSSENTROPY_LOSS(w, g)  w*ln(g) + (1-w)*ln(1 - g)
#define CROSSENTROPY_DLOSS(w, g) (w - g) / (g * (1-g))

//		d(w*ln(g) + (1-w)*ln(1-g))/dg
//	=	w/g - (1-w)/(1-g)
//	=	w(1-g)/(g*(1-g)) - g*(1 - w) / (1 - g)*g
//	= [w*(1-g) - g*(1-w)] / (g*(1 - g))
//	= [w - wg - g + gw] / [g*(1 - g)]
//	= [w - g]/[g * (1 - g)] 

//=================================================================================================
//===================================== dLOSS(g,w)/dg =============================================
//=================================================================================================

static __global__ void opti_kernel_ce_dloss(
	float * grad, float * var, float * output,
	uint total, uint ostart, uint lines, uint outs)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint line = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (out < outs && line < lines)
	{
		uint pos = line*sets*total + set*total + ostart + out;
		grad[pos] = CROSSENTROPY_DLOSS(var[pos], output[line*outs + out]);
	};
};

void CROSSENTROPY_dloss(Opti_t * opti) {
	Train_t * train = opti->train;

	kernel_ce_loss<<<dim3(KERN_DIV(train->mdl->outputs, 16), KERN_DIV(train->data->lines, 16), train->sets),dim3(16, 16, 1)>>>(
		train->_grad, train->_var, train->data->output_d,
		train->mdl->total, train->mdl->vars, train->data->lines, train->data->outputs
	);
};

//=================================================================================================
//====================================== LOSS(g,w) ================================================
//=================================================================================================

static __global__ void opit_kernel_ce_loss(
	float * grad, float * var, float * output,
	uint total, uint ostart, uint lines, uint outs)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint line = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (out < outs && line < lines)
	{
		uint pos = line*sets*total + set*total + ostart + out;
		float g = var[pos];
		float w = output[line*outs + out];
		grad[pos] = CROSSENTROPY_LOSS(g, w);
	};
};

void CROSSENTROPY_score(Opti_t * opti) {
	Train_t * train = opti->train;
	Mdl_t * mdl = train->mdl;

	uint outs = mdl->outputs;
	uint lines = train->data->lines;
	uint sets = train->sets;
	uint out_start = mdl->vars;

	opti_kernel_ce_loss<<<dim3(KERN_DIV(outs, 16), KERN_DIV(lines, 16), sets),dim3(16,16,1)>>>(
		train->_grad, train->_var, train->data->output_d,
		mdl->total, out_start, lines, outs);

	opti_kernel_sum_scores_over_lines<<<dim3(KERN_DIV(outs, 16), sets),dim3(16,1)>>>(
		train->_grad, train->_var, train->data->output_d,
		mdl->total, lines, sets, out_start, outs);

	opti_kernel_sum_scores_over_outputs<<<dim3(sets),dim3(1)>>>(
		train->_grad, opti->set_score_d,		//	<---- ??
		mdl->total, sets, output_start, outs);
};