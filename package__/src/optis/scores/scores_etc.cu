#include "hip/hip_runtime.h"
#include "package/head/optis/scores/scores_etc.cuh"

//	====== Sum all score from all lines to each output =============

//		From
//	l=0	[.......,err(o0),err(o1),err(o2),err(o3)]
//	l=1	[.......,err(o0),err(o1),err(o2),err(o3)]
//	l=2	[.......,err(o0),err(o1),err(o2),err(o3)]
//		To
//	l=0 [......., (1/lines)*(err[l0](o0)+err[l1](o0)+err[l2](o0)), (1/lines)*(err[l0](o1)+err[l1](o1)+err[l2](o1)), (1/lines)*(err[l0](o2)+err[l1](o2)+err[l2](o2)), (1/lines)*(err[l0](o3)+err[l1](o3)+err[l2](o3))]
//	l=1	[.......,err(o0),err(o1),err(o2),err(o3)]
//	l=2	[.......,err(o0),err(o1),err(o2),err(o3)]

//
//	Dessiner tout sur papier puis photocopier et mettre dans la documentation
//
//
__global__ void opti_kernel_sum_scores_over_lines(
	float * grad, float * var, float * output,
	uint total, uint lines, uint sets, uint ostart, uint outs)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = threadIdx.y;

	if (out < outs)
	{
		uint pos;
		float _sum_of_lines = 0;
		for (uint l=0; l < lines; l++) {
			_sum_of_lines += grad[l*sets*total + set*total + ostart + out];
		}
		grad[0*sets*total + set*total + ostart + out] = _sum_of_lines / lines;
	};
};

//	========= Sum all outputs to each set ===========

//		From
// set=0	l=0 [......., err[l0](o0)+err[l1](o0)+err[l2](o0), err[l0](o1)+err[l1](o1)+err[l2](o1), err[l0](o2)+err[l1](o2)+err[l2](o2), err[l0](o3)+err[l1](o3)+err[l2](o3)]
// set=1	l=0 [......., err[l0](o0)+err[l1](o0)+err[l2](o0), err[l0](o1)+err[l1](o1)+err[l2](o1), err[l0](o2)+err[l1](o2)+err[l2](o2), err[l0](o3)+err[l1](o3)+err[l2](o3)]

//		To
//	set_scores[0] = (1/(outputs))*sum(set=0 l=0 [......., err[l0](o0)+err[l1](o0)+err[l2](o0), err[l0](o1)+err[l1](o1)+err[l2](o1), err[l0](o2)+err[l1](o2)+err[l2](o2), err[l0](o3)+err[l1](o3)+err[l2](o3)])
//	set_scores[1] = (1/(outputs))*sum(set=1 l=0 [......., err[l0](o0)+err[l1](o0)+err[l2](o0), err[l0](o1)+err[l1](o1)+err[l2](o1), err[l0](o2)+err[l1](o2)+err[l2](o2), err[l0](o3)+err[l1](o3)+err[l2](o3)])

__global__ void opti_kernel_sum_scores_over_outputs(
	float * grad, float * scores,
	uint total, uint sets, uint ostart, uint outs)
{
	uint set = threadIdx.x;

	uint start = 0*sets*total + set*total + ostart + 0;
	float _sum_of_outs = 0;
	for (uint o=0; o < outs; o++) {
		_sum_of_outs += grad[start];
		start++;
	}

	scores[set] = _sum_of_lines / outs;
};