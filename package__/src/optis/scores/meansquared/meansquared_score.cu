#include "hip/hip_runtime.h"
#include "package/head/optis/scores/meansquared.cuh"

#define MEANSQUARED_LOSS(w, g) pow(g - w, 2)/2
#define MEANSQUARED_DLOSS(w, g)	g - w

//=================================================================================================
//===================================== dLOSS(g,w)/dg =============================================
//=================================================================================================

static __global__ void opti_kernel_ms_dloss(
	float * grad, float * var, float * output,
	uint total, uint ostart, uint lines, uint outs)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint line = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (out < outs && line < lines)
	{
		uint pos = line*sets*total + set*total + ostart + out;
		grad[pos] = MEANSQUARED_DLOSS(var[pos], output[line*outs + out]);
	};
};

void MEANSQUARED_dloss(Opti_t * opti) {
	Train_t * train = opti->train;

	kernel_ms_loss<<<dim3(KERN_DIV(train->mdl->outputs, 16), KERN_DIV(train->data->lines, 16), train->sets),dim3(16, 16, 1)>>>(
		train->_grad, train->_var, train->data->output_d,
		train->mdl->total, train->mdl->vars, train->data->lines, train->data->outputs
	);
};

//=================================================================================================
//====================================== LOSS(g,w) ================================================
//=================================================================================================

static __global__ void opit_kernel_ms_loss(
	float * grad, float * var, float * output,
	uint total, uint ostart, uint lines, uint outs)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint line = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (out < outs && line < lines)
	{
		uint pos = line*sets*total + set*total + ostart + out;
		float g = var[pos];
		float w = output[line*outs + out];
		grad[pos] = MEANSQUARED_LOSS(g, w);
	};
};

void MEANSQUARED_score(Opti_t * opti) {
	Train_t * train = opti->train;
	Mdl_t * mdl = train->mdl;

	uint outs = mdl->outputs;
	uint lines = train->data->lines;
	uint sets = train->sets;
	uint out_start = mdl->vars;

	opti_kernel_ms_loss<<<dim3(KERN_DIV(outs, 16), KERN_DIV(lines, 16), sets),dim3(16,16,1)>>>(
		train->_grad, train->_var, train->data->output_d,
		mdl->total, out_start, lines, outs);

	opti_kernel_sum_scores_over_lines<<<dim3(KERN_DIV(outs, 16), sets),dim3(16,1)>>>(
		train->_grad, train->_var, train->data->output_d,
		mdl->total, lines, sets, out_start, outs);

	opti_kernel_sum_scores_over_outputs<<<dim3(sets),dim3(1)>>>(
		train->_grad, opti->set_score_d,		//	<---- ??
		mdl->total, sets, output_start, outs);
};