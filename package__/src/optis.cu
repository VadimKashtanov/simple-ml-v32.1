#include "package/head/optis.cuh"

/*		-------------    Build  ------------ */
Opti_t * opti_mk(Train_t * train, uint score_algo, uint opti_algo) {
	if (score_algo >= SCORES)
		ERR("Score number %li doesn't exists. Max is %li", score_algo, SCORES - 1)
	if (opti_algo >= OPTIS)
		ERR("Opti number %li doesn't exists. Max is %li", opti_algo, OPTIS - 1)

	Opti_t * ret = (Opti_t*)malloc(sizeof(Opti_t));
	
	ret->train = train;

	//	Cpu ram arrays
	ret->set_score = (float*)malloc(sizeof(float) * train->sets);
	ret->set_rank = (uint*)malloc(sizeof(uint) * train->sets);

	//	Gpu vram arrays
	SAFE_CUDA(hipMalloc((void**)&ret->set_score_d, sizeof(float) * train->sets));
	SAFE_CUDA(hipMalloc((void**)&ret->set_rank_d, sizeof(uint) * train->sets));

	//	Algorithms
	ret->score_algo = score_algo;
	ret->opti_algo = opti_algo;

	ret->score_space = OPTI_SCORE_SPACE_MK_ARRAY[score_algo](ret);
	ret->opti_space = OPTI_OPTI_SPACE_MK_ARRAY[opti_algo](ret);

	return ret;
};

/*		-------------    Compute Score  ------------ */
bool is_sorted(float * scores, uint * podium, uint n) {
	for (uint i=1; i < n; i++)
		if (scores[podium[i-1]] > scores[podium[i]])
			return false;
	return true;
};

void opti_score(Opti_t * opti) {
	//	============== Compute Scores ============
	OPTI_COMPUTE_SCORE_ARRAY[opti->score_algo](opti);

	//	============== Compute rank ==============
	
	uint sets = opti->train->sets;

	float * scores = opti->set_scores;
	float podium[sets];	//on 0th place is the best set (ca peut tres bien etre 32, 4 ou 0)

	for (uint i=0; i < sets; i++)
		podium[i] = i;

	//Rank score
	uint c;
	while (! is_sorted(scores, podium, sets) ) {
		for (uint i=1; i < sets; i++) {
			if (scores[podium[i-1]] > scores[podium[i]]) {
				c = podium[i];
				podium[i] = podium[i-1];
				podium[i-1] = c;
			}
		}
	}

	//	On podium are sorted sets. podium[0] == id of the best set
	//	Put in set_rank. So set_rank[i] = rank of i'th set
	for (uint i=0; i < sets; i++) {
		//	The i'th place on podium is set to set_rank[podium[i]]
		opti->set_rank[podium[i]] = i; 
	}

	SAFE_CUDA(hipMemcpy(opti->set_rank_d, opti->set_rank, sizeof(uint) * sets, hipMemcpyHostToDevice));

	//	Build podium
	for (uint i=0; i < sets; i++)
		opti->podium[opti->set_rank[i]] = i;	//in set_rank sets are stored in order from 0th set to last, and each case have the position on podium
};

/*		-------------    Optimize  ------------ */
void opti_dloss(Opti_t * opti) {
	OPTI_SCORES_DLOSS_ARRAY[opti->score_algo](opti);
};

void opti_opti(Opti_t * opti) {
	OPTI_OPTIMIZE_ARRAY[opti->opti_algo](opti);
};

/*		-------------    Free structure  ------------ */
void opti_free(Opti_t * opti) {
	free(opti->scores);
	free(opti->rank);

	SAFE_CUDA(hipFree(opti->scores_d));
	SAFE_CUDA(hipFree(opti->rank_d));

	OPTI_FREE_SCORE_ARRAY[opti->score_algo](opti);
	OPTI_FREE_OPTI_ARRAY[opti->opti_algo](opti);

	free(opti);
};

//	-----------------------------------------------------------------------

//	Mk score
void* (*OPTI_SCORE_SPACE_MK_ARRAY[SCORES])(Opti_t * opti) = {
	MEANSQUARED_space_mk,
	CROSSENTROPY_space_mk
};

//	Mk opti
void* (*OPTI_OPTI_SPACE_MK_ARRAY[OPTIS])(Opti_t * opti) = {
	SGD_space_mk,
	MOMENTUM_space_mk,
	RMSPROP_space_mk,
	ADAM_space_mk,
	ADADELTA_space_mk,
	ADAMAX_space_mk
};

//	Score
void* (*OPTI_COMPUTE_SCORE_ARRAY[SCORES])(Opti_t * opti) = {
	MEANSQUARED_score,
	CROSSENTROPY_score
};

//	Score DLOSS
void (*OPTI_SCORES_DLOSS_ARRAY[SCORES])(Opti_t * opti) = {
	MEANSQUARED_dloss,
	CROSSENTROPY_dloss
};

//	Optimize
void* (*OPTI_OPTIMIZE_ARRAY[OPTIS])(Opti_t * opti) = {
	SGD_optimize,
	MOMENTUM_optimize,
	RMSPROP_optimize,
	ADAM_optimize,
	ADADELTA_optimize,
	ADAMAX_optimize
};

//	Free score
void* (*OPTI_FREE_SCORE_ARRAY[SCORES])(Opti_t * opti) = {
	MEANSQUARED_free,
	CROSSENTROPY_free
};

//	Free opti
void* (*OPTI_FREE_OPTI_ARRAY[OPTIS])(Opti_t * opti) = {
	SGD_optimize,
	MOMENTUM_free,
	RMSPROP_free,
	ADAM_free,
	ADADELTA_free,
	ADAMAX_free
};