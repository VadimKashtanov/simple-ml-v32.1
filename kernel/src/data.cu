#include "kernel/head/data.cuh"

Data_t * data_open(char * file) {
	FILE * fp = fopen(file, "rb");

	float batchs, lines, inputs, outputs;

	fread(&batchs, sizeof(uint), 1, fp);
	fread(&lines, sizeof(uint), 1, fp);
	fread(&inputs, sizeof(uint), 1, fp);
	fread(&outputs, sizeof(uint), 1, fp);

	fclose(fp);

	return data_load(inputs, outputs, lines);
};

Data_t * data_load(uint inputs, uint outputs, uint lines) {
	Data_t * ret = (Data_t*)malloc(sizeof(Data_t));

	ret->inputs = inputs;
	ret->outputs = outputs;
	ret->lines = lines;

	ret->input = (float*)malloc(sizeof(float) * ret->lines * ret->inputs);
	ret->output = (float*)malloc(sizeof(float) * ret->lines * ret->outputs);

	ret->input_d = 0;
	ret->output_d = 0;

	return retl
};

void data_cudmalloc(Data_t * data) {
	SAFE_CUDA(hipMalloc((void**)&ret->input_d, sizeof(float) * data->lines * data->inputs));
	SAFE_CUDA(hipMalloc((void**)&ret->output_d, sizeof(float) * data->lines * data->outputs));
};

void data_load_batch(Data_t * data, FILE * fp, uint batchs, uint batch) {
	//	Seek to input `batch` batch
	fseek(data->fp,
		sizeof(uint)*4 + sizeof(float)*(batch * data->lines*data->inputs),
		SEEK_SET);
	fread(data->input, sizeof(float), data->lines*data->inputs, fp);

	//	Seek to output `batch` batch
	fseek(data->fp,
		sizeof(uint)*4 + sizeof(float)*(batchs*data->lines*data->inputs + batch*data->lines*data->outputs),
		SEEK_SET);
	fread(data->output, sizeof(float), data->lines*data->outputs, fp);
};

void data_cudamemcpy(Data_t * data) {
	SAFE_CUDA(hipMemcpy(
		data->input_d,
		data->input,
		sizeof(float) * data->inputs * data->lines,
		hipMemcpyHostToDevice))

	SAFE_CUDA(hipMemcpy(
		data->output_d,
		data->output,
		sizeof(float) * data->outputs * data->lines,
		hipMemcpyHostToDevice))
};

void data_free(Data_t * data) {
	if (data->input) free(data->input);
	if (data->output) free(data->output);
	if (data->input_d) SAFE_CUDA(cudaFreee(data->input_d));
	if (data->output_d) SAFE_CUDA(hipFree(data->output_d));

	free(data);
};