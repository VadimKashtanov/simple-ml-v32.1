#include "kernel/head/use.cuh"

Use_t* use_mk(Mdl_t * mdl, Data_t * data) {
	Use_t * ret = (Use_t*)malloc(sizeof(Use_t));

	//	Dependances
	ret->mdl = mdl;
	ret->data = data;

	//	Weights
	SAFE_CUDA(hipMalloc((void**)&ret->weight_d, sizeof(float) * mdl->weights));
	SAFE_CUDA(hipMemcpy(ret->weight, mdl->weight, sizeof(float) * mdl->weights, hipMemcpyHostToDevice));

	//	Vars
	SAFE_CUDA(hipMalloc((void**)&ret->var_d, sizeof(float) * data->lines * mdl->total));
	//SAFE_CUDA(hipMemset(ret->var_d, 0, sizeof(float) * data->lines * mdl->total));	//vars have to be set and start from input

	return ret;
};

void use_set_input(Use_t * use) {
	for (uint t=0; t < use->data->lines; t++) {
		SAFE_CUDA(
			hipMemcpy(
				use->var_d + t*use->mdl->total,
				data->input_d + t*use->mdl->inputs,
				sizeof(float) * use->mdl->inputs,
				hipMemcpyHostToDevice
			)
		)
	}
};

void use_forward(Use_t * use) {
	for (uint t=0; t < use->data->lines; t++)
		for (uint i=0; i < use->mdl->insts; i++)
			INST_USE[use->mdl->id[i]](use, i, t);
};

void use_free(Use_t * use) {
	SAFE_CUDA(hipFree(use->var_d));
	SAFE_CUDA(hipFree(use->weight_d));
	free(use);
};