#include "hip/hip_runtime.h"
#include "kernel/head/train.cuh"

Train_t* mk_train(Mdl_t * mdl, Data_t * data, uint sets)
{
	Train_t * ret = (Train_t*)malloc(sizeof(Train_t));

	ret->mdl = mdl;
	ret->sets = sets;

	uint lines = data->lines;

	SAFE_CUDA(hipMalloc((void**)&ret->_weight, sizeof(float) * (mdl->weights*sets)));
	SAFE_CUDA(hipMalloc((void**)&ret->_var, sizeof(float) * (mdl->total*sets*lines)));
	SAFE_CUDA(hipMalloc((void**)&ret->_locd, sizeof(float) * (mdl->locds*sets*lines)));
	SAFE_CUDA(hipMalloc((void**)&ret->_grad, sizeof(float) * (mdl->total*sets*lines)));
	SAFE_CUDA(hipMalloc((void**)&ret->_meand, sizeof(float) * (mdl->weights*sets)));

	return ret;
};

void train_free(Train_t * train) {
	SAFE_CUDA(hipFree(train->_weight));
	SAFE_CUDA(hipFree(train->_var));
	SAFE_CUDA(hipFree(train->_locd));
	SAFE_CUDA(hipFree(train->_grad));
	SAFE_CUDA(hipFree(train->_meand));

	free(train);
};

//	---------------------------------------------------------------------------------------------

static __global__ void kernel_random_weights(uint rnd_seed, uint weights, float * _weight) {
	uint wid = threadIdx.x + blockIdx.x*blockDim.x;
	uint set = threadIdx.y + blockIdx.y*blockDim.y;
	uint pos = set*weights + wid;

	if (wid < weights)
		_weight[pos] = pseudo_randomf(rnd_seed + pos);
};

void train_random_weights(Train_t * train) {
	uint rnd_seed = rand() % 10000;

	kernel_random_weights<<<dim3(KERN_DIV(train->mdl->weights,32), train->sets), dim3(32,1)>>>(
		rnd_seed, train->mdl->weights, train->_weight);
};

//	---------------------------------------------------------------------------------------------

static __global__ void kernel_random_weights_from_mdl(uint rnd_seed, uint weights, float * _weight, float * mdl_weight_d) {
	uint wid = threadIdx.x + blockIdx.x*blockDim.x;
	uint set = threadIdx.y + blockIdx.y*blockDim.y;
	uint pos = set*weights + wid;

	if (wid < weights)
		_weight[pos] = mdl_weight_d[wid] + 0.02*(pseudo_randomf(start_seed + pos) - 0.5);//0.01*(2*(rnd()-0.5)))
};

void train_random_weights_from_mdl(Train_t * train) {
	uint rnd_seed = rand() % 10000;

	float * mdl_weights_d;
	SAFE_CUDA(hipMalloc((void**)&mdl_weights_d, sizeof(float)*train->mdl->weights));
	SAFE_CUDA(hipMemcpy(mdl_weights_d, train->mdl->weight, sizeof(float)*train->mdl->weights, hipMemcpyHostToDevice));

	kernel_random_weights_from_mdl<<<dim3(KERN_DIV(train->mdl->weights, 32), train->sets),dim3(32,1)>>>(
		start_seed, train->mdl->weights, train->_weight, mdl_weights_d);

	SAFE_CUDA(hipFree(mdl_weights_d));
};

//	---------------------------------------------------------------------------------------------

void train_cpy_ws_to_mdl(Train_t * train, uint set) {
	SAFE_CUDA(hipMemcpy(
		train->mdl->weight, train->_weight + set*train->mdl->weights,
		sizeof(float)*train->mdl->weights, hipMemcpyDeviceToHost));
};

//	---------------------------------------------------------------------------------------------

Train_t * extract_to_new_train(Train_t * old, uint amount, uint * set_id) {
	Train_t * new_train = mk_train(old->mdl, old->data, amount);
	
	uint ws = old->mdl->weights;

	for (uint s=0; s < amount; s++)
		SAFE_CUDA(hipMemcpy(new_train->_weight + s*ws, old->_weight + set_id[s]*ws, sizeof(float)*ws, hipMemcpyDeviceToDevice))

	return new_train;
};

//	---------------------------------------------------------------------------------------------

static __global__ void kernel_set_input(float * _var, float * input, uint vsize, uint sets, uint inputs) {
	/*uint kid = blockIdx.x,	\
		 time = blockIdx.y,	\	//time == line
		 set = blockIdx.z;
	_var[time*sets*vsize + set*vsize + kid] = input[time*inputs + kid];*/

	uint kid = blockIdx.x,	\
		 time = blockIdx.y;	//time == line
	for (uint set=0; set < sets; set++)
		_var[time*sets*vsize + set*vsize + kid] = input[time*inputs + kid];
};

void train_set_input(Train_t * train) {
	//kernel_set_input<<<dim3(train->mdl->inputs, train->times, train->sets),dim3(1,1,1)>>>(
	//	train->_var, train->data->input_d, train->mdl->total, train->sets, train->mdl->inputs);
	kernel_set_input<<<dim3(train->mdl->inputs, train->times),dim3(1,1)>>>(
		train->_var, train->data->input_d, train->mdl->total, train->sets, train->mdl->inputs);
};

//	---------------------------------------------------------------------------------------------

void train_null_grad_meand(Train_t * train) {
	SAFE_CUDA(hipMemset(train->_meand, 0, sizeof(float) * train->sets * train->mdl->weights))
	SAFE_CUDA(hipMemset(train->_grad, 0, sizeof(float) * train->sets * train->data->lines * train->mdl->total))
};

void train_forward(Train_t * train, uint start_seed) {
	for (uint t=0; t < train->data->lines; t++) {
		for (uint i=0; i < train->mdl->insts; i++) {
			INST_FORWARD[train->mdl->id[i]](train, i, t, start_seed);
		}
	}
};

void train_backward(Train_t * train, uint start_seed) {
	for (int t=train->data->lines-1; t >= 0; t--) {
		for (int i=train->mdl->insts-1; i >= 0; i--) {
			INST_BACKWARD[train->mdl->id[i]](train, i, t, start_seed);
		}
	}
};